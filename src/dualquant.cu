#include "hip/hip_runtime.h"
/**
 * @file cusz_dualquant.cu
 * @author Jiannan Tian
 * @brief Dual-Quantization method of cuSZ.
 * @version 0.2
 * @date 2021-01-16
 * (create) 19-09-23; (release) 2020-09-20; (rev1) 2021-01-16
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>
#include <cstddef>

#include "dualquant.cuh"
#include "metadata.hh"
#include "type_aliasing.hh"

#define tix threadIdx.x
#define tiy threadIdx.y
#define tiz threadIdx.z
#define bix blockIdx.x
#define biy blockIdx.y
#define biz blockIdx.z
#define bdx blockDim.x
#define bdy blockDim.y
#define bdz blockDim.z

namespace kernel_v2 = cusz::predictor_quantizer::v2;
namespace kernel_v3 = cusz::predictor_quantizer::v3;

// v2 ////////////////////////////////////////////////////////////

template <typename Data, typename Quant>
__global__ void kernel_v2::c_lorenzo_1d1l(lorenzo_zip ctx, Data* d, Quant* q)
{
    static const auto Block = MetadataTrait<1>::Block;
    Data(&s1df)[Block]      = *reinterpret_cast<Data(*)[Block]>(&scratch);

    auto id = bix * bdx + tix;

    if (id < ctx.d0) {
        // prequant (fp presence)
        s1df[tix] = round(d[id] * ctx.ebx2_r);
        __syncthreads();  // necessary to ensure correctness
        // postquant
        Data pred = tix == 0 ? 0 : s1df[tix - 1];
        __syncthreads();

        Data delta       = s1df[tix] - pred;
        bool quantizable = fabs(delta) < ctx.radius;
        Data candidate   = delta + ctx.radius;
        d[id]            = (1 - quantizable) * candidate;  // output; reuse data for outlier
        q[id]            = quantizable * static_cast<Quant>(candidate);
    }
}

template <typename Data, typename Quant>
__global__ void kernel_v2::x_lorenzo_1d1l(lorenzo_unzip ctx, Data* xd, Data* outlier, Quant* q)
{
    static const auto Block = MetadataTrait<1>::Block;

    auto b0 = bdx * bix + tix;

    if (b0 >= ctx.nblk0) return;

    auto _idx0 = b0 * Block;

    for (auto i0 = 0; i0 < Block; i0++) {
        auto id = _idx0 + i0;
        if (id >= ctx.d0) continue;
        Data pred = id < _idx0 + 1 ? 0 : xd[id - 1];
        xd[id]    = (q[id] == 0 ? outlier[id] : static_cast<Data>(q[id])) + pred - static_cast<Data>(ctx.radius);
    }
    for (auto i0 = 0; i0 < Block; i0++) {
        size_t id = _idx0 + i0;
        if (id >= ctx.d0) continue;
        xd[id] *= ctx.ebx2;
    }
    // end of body //
}

template __global__ void kernel_v2::c_lorenzo_1d1l<FP4, UI1>(lorenzo_zip, FP4*, UI1*);
template __global__ void kernel_v2::c_lorenzo_1d1l<FP4, UI2>(lorenzo_zip, FP4*, UI2*);

template __global__ void kernel_v2::x_lorenzo_1d1l<FP4, UI1>(lorenzo_unzip, FP4*, FP4*, UI1*);
template __global__ void kernel_v2::x_lorenzo_1d1l<FP4, UI2>(lorenzo_unzip, FP4*, FP4*, UI2*);

// v3 ////////////////////////////////////////////////////////////

template <typename Data, typename Quant>
__global__ void kernel_v3::c_lorenzo_2d1l(lorenzo_zip ctx, Data* d, Quant* q)
{
    static const auto Block   = MetadataTrait<2>::Block;
    Data(&s2df)[Block][Block] = *reinterpret_cast<Data(*)[Block][Block]>(&scratch);

    auto y = tiy, x = tix;
    auto gi1 = biy * bdy + y, gi0 = bix * bdx + x;

    if (gi0 < ctx.d0 and gi1 < ctx.d1) {
        size_t id = gi0 + gi1 * ctx.stride1;  // low to high dim, inner to outer

        // prequant (fp presence)
        s2df[y][x] = round(d[id] * ctx.ebx2_r);
        __syncthreads();  // necessary to ensure correctness

        Data delta       = s2df[y][x] - ((x > 0 ? s2df[y][x - 1] : 0) +                // dist=1
                                   (y > 0 ? s2df[y - 1][x] : 0) -                // dist=1
                                   (x > 0 and y > 0 ? s2df[y - 1][x - 1] : 0));  // dist=2
        bool quantizable = fabs(delta) < ctx.radius;
        Data candidate   = delta + ctx.radius;
        d[id]            = (1 - quantizable) * candidate;  // output; reuse data for outlier
        q[id]            = quantizable * static_cast<Quant>(candidate);
    }
}

template <typename Data, typename Quant>
__global__ void kernel_v3::c_lorenzo_3d1l(lorenzo_zip ctx, Data* d, Quant* q)
{
    static const auto Block          = MetadataTrait<3>::Block;
    Data(&s3df)[Block][Block][Block] = *reinterpret_cast<Data(*)[Block][Block][Block]>(&scratch);

    auto z = tiz, y = tiy, x = tix;
    auto gi2 = biz * bdz + z, gi1 = biy * bdy + y, gi0 = bix * bdx + x;

    if (gi0 < ctx.d0 and gi1 < ctx.d1 and gi2 < ctx.d2) {
        size_t id = gi0 + gi1 * ctx.stride1 + gi2 * ctx.stride2;  // low to high in dim, inner to outer

        // prequant (fp presence)
        s3df[z][y][x] = round(d[id] * ctx.ebx2_r);
        __syncthreads();  // necessary to ensure correctness

        Data delta       = s3df[z][y][x] - ((z > 0 and y > 0 and x > 0 ? s3df[z - 1][y - 1][x - 1] : 0)  // dist=3
                                      - (y > 0 and x > 0 ? s3df[z][y - 1][x - 1] : 0)              // dist=2
                                      - (z > 0 and x > 0 ? s3df[z - 1][y][x - 1] : 0)              //
                                      - (z > 0 and y > 0 ? s3df[z - 1][y - 1][x] : 0)              //
                                      + (x > 0 ? s3df[z][y][x - 1] : 0)                            // dist=1
                                      + (y > 0 ? s3df[z][y - 1][x] : 0)                            //
                                      + (z > 0 ? s3df[z - 1][y][x] : 0));                          //
        bool quantizable = fabs(delta) < ctx.radius;
        Data candidate   = delta + ctx.radius;
        d[id]            = (1 - quantizable) * candidate;  // output; reuse data for outlier
        q[id]            = quantizable * static_cast<Quant>(candidate);
    }
}

template <typename Data, typename Quant>
__global__ void kernel_v3::x_lorenzo_2d1l(lorenzo_unzip ctx, Data* xd, Data* outlier, Quant* q)
{
    static const auto Block = MetadataTrait<2>::Block;

    Data s[Block][Block];  // try not use shared memory first
    memset(s, 0, Block * Block * sizeof(Data));

    auto b1 = bdy * biy + tiy, b0 = bdx * bix + tix;

    if (b1 >= ctx.nblk1 or b0 >= ctx.nblk0) return;

    auto _idx1 = b1 * Block, _idx0 = b0 * Block;

    for (auto i1 = 0; i1 < Block; i1++) {
        for (auto i0 = 0; i0 < Block; i0++) {
            auto gi1 = _idx1 + i1, gi0 = _idx0 + i0;

            if (gi1 >= ctx.d1 or gi0 >= ctx.d0) continue;
            size_t id   = gi0 + gi1 * ctx.stride1;
            Data   pred = (i1 > 0 ? s[i1 - 1][i0] : 0)  //
                        + (i0 > 0 ? s[i1][i0 - 1] : 0)  //
                        - (i1 > 0 and i0 > 0 ? s[i1 - 1][i0 - 1] : 0);
            s[i1][i0] = (q[id] == 0 ? outlier[id] : static_cast<Data>(q[id])) + pred - static_cast<Data>(ctx.radius);
            xd[id]    = s[i1][i0] * ctx.ebx2;
        }
    }
    // end of body //
}

template <typename Data, typename Quant>
__global__ void kernel_v3::x_lorenzo_3d1l(lorenzo_unzip ctx, Data* xd, Data* outlier, Quant* q)
{
    static const auto Block = MetadataTrait<3>::Block;

    Data s[Block][Block][Block];
    memset(s, 0, Block * Block * Block * sizeof(Data));

    auto b2 = bdz * biz + tiz, b1 = bdy * biy + tiy, b0 = bdx * bix + tix;

    if (b2 >= ctx.nblk2 or b1 >= ctx.nblk1 or b0 >= ctx.nblk0) return;

    auto _idx2 = b2 * Block, _idx1 = b1 * Block, _idx0 = b0 * Block;

    for (auto i2 = 0; i2 < Block; i2++) {
        for (auto i1 = 0; i1 < Block; i1++) {
            for (auto i0 = 0; i0 < Block; i0++) {
                auto gi2 = _idx2 + i2, gi1 = _idx1 + i1, gi0 = _idx0 + i0;

                if (gi2 >= ctx.d2 or gi1 >= ctx.d1 or gi0 >= ctx.d0) continue;
                size_t id = gi0 + gi1 * ctx.stride1 + gi2 * ctx.stride2;

                Data pred = (i2 > 0 and i1 > 0 and i0 > 0 ? s[i2 - 1][i1 - 1][i0 - 1] : 0)  // dist=3
                            - (i1 > 0 and i0 > 0 ? s[i2][i1 - 1][i0 - 1] : 0)               // dist=2
                            - (i2 > 0 and i0 > 0 ? s[i2 - 1][i1][i0 - 1] : 0)               //
                            - (i2 > 0 and i1 > 0 ? s[i2 - 1][i1 - 1][i0] : 0)               //
                            + (i0 > 0 ? s[i2][i1][i0 - 1] : 0)                              // dist=1
                            + (i1 > 0 ? s[i2][i1 - 1][i0] : 0)                              //
                            + (i2 > 0 ? s[i2 - 1][i1][i0] : 0);                             //
                s[i2][i1][i0] =
                    (q[id] == 0 ? outlier[id] : static_cast<Data>(q[id])) + pred - static_cast<Data>(ctx.radius);
                xd[id] = s[i2][i1][i0] * ctx.ebx2;
            }
        }
    }
}

template __global__ void kernel_v3::c_lorenzo_2d1l<FP4, UI1>(lorenzo_zip, FP4*, UI1*);
template __global__ void kernel_v3::c_lorenzo_2d1l<FP4, UI2>(lorenzo_zip, FP4*, UI2*);
template __global__ void kernel_v3::c_lorenzo_3d1l<FP4, UI1>(lorenzo_zip, FP4*, UI1*);
template __global__ void kernel_v3::c_lorenzo_3d1l<FP4, UI2>(lorenzo_zip, FP4*, UI2*);

template __global__ void kernel_v3::x_lorenzo_2d1l<FP4, UI1>(lorenzo_unzip, FP4*, FP4*, UI1*);
template __global__ void kernel_v3::x_lorenzo_2d1l<FP4, UI2>(lorenzo_unzip, FP4*, FP4*, UI2*);
template __global__ void kernel_v3::x_lorenzo_3d1l<FP4, UI1>(lorenzo_unzip, FP4*, FP4*, UI1*);
template __global__ void kernel_v3::x_lorenzo_3d1l<FP4, UI2>(lorenzo_unzip, FP4*, FP4*, UI2*);

namespace kernel_fm = cusz::predictor_quantizer::fine_massive;

template <typename Data, typename Quant>
__global__ void kernel_fm::x_lorenzo_1d1l(lorenzo_unzip ctx, Data* data, Data* outlier, Quant* q)
{
    static const auto Block = MetadataTrait<1>::Block;
    Data(&buffer)[Block]    = *reinterpret_cast<Data(*)[Block]>(&scratch);

    auto id     = bix * bdx + tix;
    auto radius = static_cast<Data>(ctx.radius);

    if (id < ctx.d0)
        buffer[tix] = outlier[id] + static_cast<Data>(q[id]) - radius;  // fuse
    else
        buffer[tix] = 0;
    __syncthreads();

    for (auto d = 1; d < Block; d *= 2) {
        Data n = 0;
        if (tix >= d) n = buffer[tix - d];  // like __shfl_up_sync(0x1f, var, d); warp_sync
        __syncthreads();
        if (tix >= d) buffer[tix] += n;
        __syncthreads();
    }

    if (id < ctx.d0) { data[id] = buffer[tix] * ctx.ebx2; }
    __syncthreads();
}

template <typename Data, typename Quant>
__global__ void kernel_fm::x_lorenzo_2d1l(lorenzo_unzip ctx, Data* data, Data* outlier, Quant* q)
{
    static const auto Block     = MetadataTrait<2>::Block;
    Data(&buffer)[Block][Block] = *reinterpret_cast<Data(*)[Block][Block]>(&scratch);

    auto   gi1 = biy * bdy + tiy, gi0 = bix * bdx + tix;
    size_t id     = gi0 + gi1 * ctx.stride1;
    auto   radius = static_cast<Data>(ctx.radius);

    if (gi0 < ctx.d0 and gi1 < ctx.d1)
        buffer[tiy][tix] = outlier[id] + static_cast<Data>(q[id]) - radius;  // fuse
    else
        buffer[tiy][tix] = 0;
    __syncthreads();

    for (auto d = 1; d < Block; d *= 2) {
        Data n = 0;
        if (tix >= d) n = buffer[tiy][tix - d];
        __syncthreads();
        if (tix >= d) buffer[tiy][tix] += n;
        __syncthreads();
    }

    for (auto d = 1; d < Block; d *= 2) {
        Data n = 0;
        if (tiy >= d) n = buffer[tiy - d][tix];
        __syncthreads();
        if (tiy >= d) buffer[tiy][tix] += n;
        __syncthreads();
    }

    if (gi0 < ctx.d0 and gi1 < ctx.d1) { data[id] = buffer[tiy][tix] * ctx.ebx2; }
    __syncthreads();
}

template <typename Data, typename Quant>
__global__ void kernel_fm::x_lorenzo_3d1l(lorenzo_unzip ctx, Data* data, Data* outlier, Quant* q)
{
    static const auto Block            = MetadataTrait<3>::Block;
    Data(&buffer)[Block][Block][Block] = *reinterpret_cast<Data(*)[Block][Block][Block]>(&scratch);

    auto   gi2 = biz * bdz + tiz, gi1 = biy * bdy + tiy, gi0 = bix * bdx + tix;
    size_t id     = gi0 + gi1 * ctx.stride1 + gi2 * ctx.stride2;  // low to high in dim, inner to outer
    auto   radius = static_cast<Data>(ctx.radius);

    if (gi0 < ctx.d0 and gi1 < ctx.d1 and gi2 < ctx.d2)
        buffer[tiz][tiy][tix] = outlier[id] + static_cast<Data>(q[id]) - radius;  // id
    else
        buffer[tiz][tiy][tix] = 0;
    __syncthreads();

    for (auto d = 1; d < Block; d *= 2) {
        Data n = 0;
        if (tix >= d) n = buffer[tiz][tiy][tix - d];
        __syncthreads();
        if (tix >= d) buffer[tiz][tiy][tix] += n;
        __syncthreads();
    }

    for (auto d = 1; d < Block; d *= 2) {
        Data n = 0;
        if (tiy >= d) n = buffer[tiz][tiy - d][tix];
        __syncthreads();
        if (tiy >= d) buffer[tiz][tiy][tix] += n;
        __syncthreads();
    }

    for (auto d = 1; d < Block; d *= 2) {
        Data n = 0;
        if (tiz >= d) n = buffer[tiz - d][tiy][tix];
        __syncthreads();
        if (tiz >= d) buffer[tiz][tiy][tix] += n;
        __syncthreads();
    }

    if (gi0 < ctx.d0 and gi1 < ctx.d1 and gi2 < ctx.d2) { data[id] = buffer[tiz][tiy][tix] * ctx.ebx2; }
    __syncthreads();
}

template __global__ void kernel_fm::x_lorenzo_1d1l<FP4, UI1>(lorenzo_unzip, FP4*, FP4*, UI1*);
template __global__ void kernel_fm::x_lorenzo_1d1l<FP4, UI2>(lorenzo_unzip, FP4*, FP4*, UI2*);
template __global__ void kernel_fm::x_lorenzo_2d1l<FP4, UI1>(lorenzo_unzip, FP4*, FP4*, UI1*);
template __global__ void kernel_fm::x_lorenzo_2d1l<FP4, UI2>(lorenzo_unzip, FP4*, FP4*, UI2*);
template __global__ void kernel_fm::x_lorenzo_3d1l<FP4, UI1>(lorenzo_unzip, FP4*, FP4*, UI1*);
template __global__ void kernel_fm::x_lorenzo_3d1l<FP4, UI2>(lorenzo_unzip, FP4*, FP4*, UI2*);