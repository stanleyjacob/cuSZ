#include "hip/hip_runtime.h"
/**
 * @file cusz_dryrun.cu
 * @author Jiannan Tian
 * @brief cuSZ dryrun mode, checking data quality from lossy compression.
 * @version 0.1
 * @date 2020-09-21
 * Created on 2020-05-14
 *
 * @copyright Copyright (c) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <iostream>
#include <string>
#include "cuda_mem.cuh"
#include "cusz_dryrun.cuh"
#include "cusz_dualquant.cuh"
#include "format.hh"
#include "io.hh"
#include "metadata.hh"
#include "verify.hh"

using std::cerr;
using std::cout;
using std::endl;
using std::string;

const int DIM0 = 0;
const int DIM1 = 1;
const int DIM2 = 2;
// const int DIM3   = 3;
const int nBLK0 = 4;
const int nBLK1 = 5;
const int nBLK2 = 6;
// const int nBLK3  = 7;
const int nDIM = 8;
const int LEN  = 12;
// const int CAP    = 13;
// const int RADIUS = 14;

const size_t EB = 0;
// const size_t EBr    = 1;
const size_t EBx2   = 2;
const size_t EBx2_r = 3;

const int B_1d = 32;
const int B_2d = 16;
const int B_3d = 8;

template <int Block, typename Data>
__global__ void cusz::dryrun::lorenzo_1d1l(struct Metadata<Block>* m, Data* d)
{
    auto id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= m->d0) return;
    d[id] = round(d[id] * m->ebx2_r) * m->ebx2;  // prequant
}

template <int Block, typename Data>
__global__ void cusz::dryrun::lorenzo_2d1l(struct Metadata<Block>* m, Data* d)
{
    auto   y   = threadIdx.y;
    auto   x   = threadIdx.x;
    auto   gi1 = blockIdx.y * blockDim.y + y;
    auto   gi0 = blockIdx.x * blockDim.x + x;
    size_t id  = gi0 + gi1 * m->stride1;  // low to high dim, inner to outer
    if (gi0 >= m->d0 or gi1 >= m->d1) return;
    d[id] = round(d[id] * m->ebx2_r) * m->ebx2;  // prequant
}

template <int Block, typename Data>
__global__ void cusz::dryrun::lorenzo_3d1l(struct Metadata<Block>* m, Data* d)
{
    auto   gi2 = blockIdx.z * blockDim.z + threadIdx.z;
    auto   gi1 = blockIdx.y * blockDim.y + threadIdx.y;
    auto   gi0 = blockIdx.x * blockDim.x + threadIdx.x;
    size_t id  = gi0 + gi1 * m->stride1 + gi2 * m->stride2;  // low to high in dim, inner to outer
    if (gi0 >= m->d0 or gi1 >= m->d1 or gi2 >= m->d2) return;
    d[id] = round(d[id] * m->ebx2_r) * m->ebx2;  // prequant
}

////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////

/**
 * @deprecated 0.1.1 or higher
 */
template <typename Data>
__global__ void cusz::dryrun::lorenzo_1d1l(Data* data, size_t* dims_L16, double* ebs_L4)
{
    auto id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= dims_L16[DIM0]) return;
    data[id] = round(data[id] * ebs_L4[EBx2_r]) * ebs_L4[EBx2];  // prequantization
}

/**
 * @deprecated 0.1.1 or higher
 */
template <typename Data>
__global__ void cusz::dryrun::lorenzo_2d1l(Data* data, size_t* dims_L16, double* ebs_L4)
{
    auto   y   = threadIdx.y;
    auto   x   = threadIdx.x;
    auto   gi1 = blockIdx.y * blockDim.y + y;
    auto   gi0 = blockIdx.x * blockDim.x + x;
    size_t id  = gi0 + gi1 * dims_L16[DIM0];  // low to high dim, inner to outer
    if (gi0 >= dims_L16[DIM0] or gi1 >= dims_L16[DIM1]) return;
    data[id] = round(data[id] * ebs_L4[EBx2_r]) * ebs_L4[EBx2];  // prequantization
}

/**
 * @deprecated 0.1.1 or higher
 */
template <typename Data>
__global__ void cusz::dryrun::lorenzo_3d1l(Data* data, size_t* dims_L16, double* ebs_L4)
{
    auto   gi2 = blockIdx.z * blockDim.z + threadIdx.z;
    auto   gi1 = blockIdx.y * blockDim.y + threadIdx.y;
    auto   gi0 = blockIdx.x * blockDim.x + threadIdx.x;
    size_t id =
        gi0 + gi1 * dims_L16[DIM0] + gi2 * dims_L16[DIM0] * dims_L16[DIM1];  // low to high in dim, inner to outer
    if (gi0 >= dims_L16[DIM0] or gi1 >= dims_L16[DIM1] or gi2 >= dims_L16[DIM2]) return;
    data[id] = round(data[id] * ebs_L4[EBx2_r]) * ebs_L4[EBx2];  // prequantization
}

template <typename Data>
void cusz::interface::DryRun(Data* d, Data* d_d, string fi, size_t* dims, double* ebs)
{
    cout << log_info << "Entering dry-run mode..." << endl;
    auto len        = dims[LEN];
    auto d_dims_L16 = mem::CreateDeviceSpaceAndMemcpyFromHost(dims, 16);
    auto d_ebs_L4   = mem::CreateDeviceSpaceAndMemcpyFromHost(ebs, 4);

    if (dims[nDIM] == 1) {
        dim3 blockNum(dims[nBLK0]);
        dim3 threadNum(B_1d);
        cusz::dryrun::lorenzo_1d1l<Data><<<blockNum, threadNum>>>(d_d, d_dims_L16, d_ebs_L4);
    }
    else if (dims[nDIM] == 2) {
        dim3 blockNum(dims[nBLK0], dims[nBLK1]);
        dim3 threadNum(B_2d, B_2d);
        cusz::dryrun::lorenzo_2d1l<Data><<<blockNum, threadNum>>>(d_d, d_dims_L16, d_ebs_L4);
    }
    else if (dims[nDIM] == 3) {
        dim3 blockNum(dims[nBLK0], dims[nBLK1], dims[nBLK2]);
        dim3 threadNum(B_3d, B_3d, B_3d);
        cusz::dryrun::lorenzo_3d1l<Data><<<blockNum, threadNum>>>(d_d, d_dims_L16, d_ebs_L4);
    }
    hipDeviceSynchronize();
    hipMemcpy(d, d_d, len * sizeof(Data), hipMemcpyDeviceToHost);

    auto d2 = io::ReadBinaryToNewArray<Data>(fi, len);
    // CR is not valid in dry run
    analysis::VerifyData<Data>(d, d2, len, false, ebs[EB], 0);
    cout << log_info << "Dry-run finished, exit..." << endl;
    delete[] d;
    delete[] d2;
    hipFree(d_d);
    hipFree(d_dims_L16);
    hipFree(d_ebs_L4);
}

template __global__ void cusz::dryrun::lorenzo_1d1l<float>(float*, size_t*, double*);
template __global__ void cusz::dryrun::lorenzo_2d1l<float>(float*, size_t*, double*);
template __global__ void cusz::dryrun::lorenzo_3d1l<float>(float*, size_t*, double*);

template void cusz::interface::DryRun<float>(float* d, float* d_d, string fi, size_t* dims, double* ebs);
/*
template void cusz::interface::DryRun<double>(double* d, double* d_d, string fi, size_t* dims, double* ebs);
template void cusz::interface::DryRun<char>(char* d, char* d_d, string fi, size_t* dims, double* ebs);
template void cusz::interface::DryRun<short>(short* d, short* d_d, string fi, size_t* dims, double* ebs);
template void cusz::interface::DryRun<int>(int* d, int* d_d, string fi, size_t* dims, double* ebs);
template void cusz::interface::DryRun<long>(long* d, long* d_d, string fi, size_t* dims, double* ebs);
template void cusz::interface::DryRun<long long>(long long* d, long long* d_d, string fi, size_t* dims, double* ebs);
 */
