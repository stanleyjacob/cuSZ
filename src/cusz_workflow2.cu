#include "hip/hip_runtime.h"
/**
 * @file cusz_workflow2.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.1.1
 * @date 2020-09-23
 *
 * Copyright (c) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 *
 */

#include <string>

#include "argparse2_cusz.hh"
#include "cuda_error_handling.cuh"
#include "cuda_mem.cuh"
#include "cusz_workflow.cuh"
#include "cusz_workflow2.cuh"
#include "gather_scatter.cuh"
#include "huffman_workflow.cuh"
#include "io.hh"
#include "lorenzo_trait.cu"
// #include "lorenzo_trait.cuh"
#include "metadata.hh"
#include "type_trait.hh"

template <int ndim, int Block, typename Data, int QuantByte, int HuffByte>
void cusz::interface::Compress2(cuszContext* ctx, struct Metadata<Block>* m)
{
    typedef struct Metadata<Block>                metadata_t;
    typedef typename QuantTrait<QuantByte>::Quant Quant;
    typedef typename HuffTrait<HuffByte>::Huff    Huff;

    string fo_zip     = ctx->get_fname() + ".sza";
    string fo_q       = ctx->get_fname() + ".b" + std::to_string(QuantByte * 8);
    string fo_outlier = ctx->get_fname() + ".b" + std::to_string(QuantByte * 8) + "outlier";

    auto M   = cusz::impl::GetEdgeOfReinterpretedSquare(m->len);
    auto MxM = M * M;

    auto data = new Data[MxM]();
    io::ReadBinaryFile<Data>(ctx->get_fname(), data, m->len);
    auto d_data = mem::CreateDeviceSpaceAndMemcpyFromHost(data, MxM);

    if (ctx->wf_dryrun) {
        if (ndim == 1)
            ::dryrun::Lorenzo_nd1l<1>::Call<Block, Data, Quant>(m, d_data);
        else if (ndim == 2)
            ::dryrun::Lorenzo_nd1l<2>::Call<Block, Data, Quant>(m, d_data);
        else if (ndim == 3)
            ::dryrun::Lorenzo_nd1l<3>::Call<Block, Data, Quant>(m, d_data);
        delete[] data, hipFree(d_data), exit(0);
    }

    metadata_t* d_m;
    hipMalloc((void**)&d_m, sizeof(metadata_t));
    hipMemcpy(d_m, m, sizeof(metadata_t), hipMemcpyHostToDevice);

    auto d_q = mem::CreateCUDASpace<Quant>(m->len);

    {  // Lorenzo
        void*  args[] = {&d_m, &d_data, &d_q};
        dim3   grid_dim(m->nb0, m->nb1, m->nb2), block_dim(m->b0, m->b1, m->b2);
        size_t cache_size = Block;
        for (auto i = 0; i < ndim - 1; i++) cache_size *= Block;

        if (ndim == 1)  // compile time?
            hipLaunchKernel(
                (void*)zip::Lorenzo_nd1l<1>::Call<Block, Data, Quant>,  //
                grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);
        if (ndim == 2)  // compile time?
            hipLaunchKernel(
                (void*)zip::Lorenzo_nd1l<2>::Call<Block, Data, Quant>,  //
                grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);
        else if (ndim == 3)  // compile time?
            hipLaunchKernel(
                (void*)zip::Lorenzo_nd1l<3>::Call<Block, Data, Quant>,  //
                grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);

        // goal:
        // hipLaunchKernel(
        //     (void*)zip::Lorenzo_nd1l<ndim>::Call<Block, Data, Quant>,  //
        //     grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);

        CHECK_CUDA(hipDeviceSynchronize());
    }

    if (ctx->skip_huff) {
        auto q = mem::CreateHostSpaceAndMemcpyFromDevice(d_q, m->len);
        io::WriteBinaryFile(q, m->len, &fo_q);
        // TODO log
        delete[] q, delete[] data, hipFree(d_q), hipFree(d_data), exit(0);
    }

    // handle outlier
    ::cusz::impl::PruneGatherAsCSR(d_data, MxM, M /*lda*/, M /*m*/, M /*n*/, m->nnz, &fo_outlier);

    // TODO handle metadata
    std::tie(m->n_bits, m->n_uint, m->huff_metadata_size) =
        lossless::interface::HuffmanEncode<Quant, Huff>(fo_q, d_q, m->len, ctx->h_chunksize, m->cap);

    cout << log_info << "Compression finished, saved Huffman encoded quant.code.\n" << endl;

    delete[] data;
    hipFree(d_data);
}

template <int ndim, int Block, typename Data, int QuantByte, int HuffByte>
void cusz::interface::Decompress2(cuszContext* ctx, struct Metadata<Block>* m)
{
    typedef struct Metadata<Block>                metadata_t;  // instead of `typename`
    typedef typename QuantTrait<QuantByte>::Quant Quant;
    typedef typename HuffTrait<HuffByte>::Huff    Huff;

    string fo_x       = ctx->get_fname() + ".szx";
    string fi_qbase   = ctx->get_fname() + ".b" + std::to_string(QuantByte * 8);
    string fi_outlier = fi_qbase + ".outlier";

    auto M   = ::cusz::impl::GetEdgeOfReinterpretedSquare(m->len);
    auto MxM = M * M;

    Quant* xq;
    // step 1: read from filesystem or do Huffman decoding to get quant code
    if (ctx->skip_huff) { xq = io::ReadBinaryFile<Quant>(fi_qbase, m->len); }
    else {
        xq = ::lossless::interface::HuffmanDecode<Quant, Huff>(
            fi_qbase, m->len, ctx->h_chunksize, m->total_uint, m->cap);
        if (ctx->verify_huffman) cusz::impl::VerifyHuffman<Data, Quant>(ctx, m, xq);
    }
}

template <int ndim, int Block, typename Data, typename Quant>
void cusz::impl::VerifyHuffman(cuszContext* ctx, struct Metadata<Block>* m, Quant* xq)
{
    typedef struct Metadata<Block> metadata_t;
    // TODO error handling from invalid read
    // cout << log_info << "Redo PdQ just to get quantization dump." << endl;

    auto  data   = io::ReadBinaryFile<Data>(ctx->get_fname(), m->len);
    Data* d_data = mem::CreateDeviceSpaceAndMemcpyFromHost(data, m->len);
    auto  d_q    = mem::CreateCUDASpace<Quant>(m->len);

    metadata_t* d_m;
    hipMalloc((void**)&d_m, sizeof(metadata_t));
    hipMemcpy(d_m, m, sizeof(metadata_t), hipMemcpyHostToDevice);

    {  // Lorenzo
        void*  args[] = {&d_m, &d_data, &d_q};
        dim3   grid_dim(m->nb0, m->nb1, m->nb2), block_dim(m->b0, m->b1, m->b2);
        size_t cache_size = Block;
        for (auto i = 0; i < m->ndim - 1; i++) cache_size *= Block;

        if (ndim == 1)  // compile time?
            hipLaunchKernel(
                (void*)zip::Lorenzo_nd1l<1>::Call<Block, Data, Quant>,  //
                grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);
        if (ndim == 3)  // compile time?
            hipLaunchKernel(
                (void*)zip::Lorenzo_nd1l<2>::Call<Block, Data, Quant>,  //
                grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);
        else if (ndim == 2)  // compile time?
            hipLaunchKernel(
                (void*)zip::Lorenzo_nd1l<3>::Call<Block, Data, Quant>,  //
                grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);

        CHECK_CUDA(hipDeviceSynchronize());
    }

    auto len    = m->len;
    auto veri_q = mem::CreateHostSpaceAndMemcpyFromDevice(d_q, len);

    auto count = 0;
    for (auto i = 0; i < len; i++)
        if (xq[i] != veri_q[i]) count++;
    if (count != 0)
        cerr << log_err << "percentage of not being equal: " << count / (1.0 * len) << "\n";
    else
        cout << log_info << "Decoded correctly." << endl;

    if (count != 0) {
        auto n_chunk = (len - 1) / ctx->h_chunksize + 1;
        for (auto c = 0; c < n_chunk; c++) {
            auto chunk_id_printed   = false;
            auto prev_point_printed = false;
            for (auto i = 0; i < ctx->h_chunksize; i++) {
                auto idx = i + c * ctx->h_chunksize;
                if (idx >= len) break;
                if (xq[idx] != xq[idx]) {
                    if (not chunk_id_printed) {
                        cerr << "chunk id: " << c << "\t";
                        cerr << "start@ " << c * ctx->h_chunksize << "\tend@ " << (c + 1) * ctx->h_chunksize - 1
                             << endl;
                        chunk_id_printed = true;
                    }
                    if (not prev_point_printed) {
                        if (idx != c * ctx->h_chunksize) {  // not first point
                            cerr << "PREV-idx:" << idx - 1 << "\t" << xq[idx - 1] << "\t" << xq[idx - 1] << endl;
                        }
                        else {
                            cerr << "wrong at first point!" << endl;
                        }
                        prev_point_printed = true;
                    }
                    cerr << "idx:" << idx << "\tdecoded: " << xq[idx] << "\tori: " << xq[idx] << endl;
                }
            }
        }
    }

    hipFree(d_q), hipFree(d_data);
    delete[] veri_q, delete[] data;
    // end of if count
}