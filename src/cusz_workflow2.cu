#include "hip/hip_runtime.h"
/**
 * @file cusz_workflow2.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.1.1
 * @date 2020-09-23
 *
 * Copyright (c) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 *
 */

#include <memory>
#include <string>

#include "argparse2_cusz.hh"
#include "cuda_error_handling.cuh"
#include "cuda_mem.cuh"
// #include "cusz_workflow.cuh"
#include "cusz_dualquant.cuh"
#include "cusz_workflow.cuh"
#include "cusz_workflow2.cuh"
#include "format.hh"
#include "gather_scatter.cuh"
#include "huffman_workflow.cuh"
#include "io.hh"
#include "lorenzo_trait.cu"
// #include "lorenzo_trait.cuh"
#include "metadata.hh"
#include "type_trait.hh"

typedef cuszContext ctx_t;

template <int ndim, typename Data, int QuantByte, int HuffByte>
void cusz::interface::Compress2(ctx_t* ctx, typename MetadataTrait<ndim>::metadata_t* m)
{
    typedef typename MetadataTrait<ndim>::metadata_t metadata_t;
    typedef typename QuantTrait<QuantByte>::Quant    Quant;
    typedef typename HuffTrait<HuffByte>::Huff       Huff;

    static const int Block = MetadataTrait<ndim>::Block;

    string fo_zip     = ctx->get_fname() + ".sza";
    string fo_q       = ctx->get_fname() + ".b" + std::to_string(QuantByte * 8);
    string fo_outlier = ctx->get_fname() + ".b" + std::to_string(QuantByte * 8) + "outlier";

    auto M   = cusz::impl::GetEdgeOfReinterpretedSquare(m->len);
    auto MxM = M * M;

    // auto data = new Data[MxM]();
    auto data = std::unique_ptr<Data[]>(new Data[MxM]());

    io::ReadBinaryToArray<Data>(ctx->get_fname(), data.get(), m->len);
    auto d_data = mem::CreateDeviceSpaceAndMemcpyFromHost(data.get(), MxM);

    metadata_t* d_m;
    hipMalloc((void**)&d_m, sizeof(metadata_t));
    hipMemcpy(d_m, m, sizeof(metadata_t), hipMemcpyHostToDevice);

    if (ctx->wf_dryrun) {
        void* args[] = {&d_m, &d_data};
        dim3  grid_dim(m->nb0, m->nb1, m->nb2), block_dim(m->b, m->b, m->b);
#if __cplusplus >= 201703L
        hipLaunchKernel(
            (void*)dryrun::Lorenzo_nd1l<ndim>::Call<Block, Data>,  //
            grid_dim, block_dim, args, 0, nullptr);
#elif __cplusplus >= 201402L
        if (ndim == 1)
            hipLaunchKernel((void*)cusz::dryrun::lorenzo_1d1l<Block, Data>, grid_dim, block_dim, args, 0, nullptr);
        else if (ndim == 2)
            hipLaunchKernel((void*)cusz::dryrun::lorenzo_2d1l<Block, Data>, grid_dim, block_dim, args, 0, nullptr);
        else if (ndim == 3)
            hipLaunchKernel((void*)cusz::dryrun::lorenzo_3d1l<Block, Data>, grid_dim, block_dim, args, 0, nullptr);
#endif
        goto COMPRESS_END;
    }

    if (ctx->wf_zip) {
        auto d_q = mem::CreateCUDASpace<Quant>(m->len);

        {  // Lorenzo
            void*  args[] = {&d_m, &d_data, &d_q};
            dim3   grid_dim(m->nb0, m->nb1, m->nb2), block_dim(m->b, m->b, m->b);
            size_t cache_size = Block;
            for (auto i = 0; i < ndim - 1; i++) cache_size *= Block;

#if __cplusplus >= 201703L
            hipLaunchKernel(
                (void*)zip::Lorenzo_nd1l<ndim>::Call<Block, Data, Quant>,  //
                grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);
#elif __cplusplus >= 201402L
            if (ndim == 1)  // compile time?
                hipLaunchKernel(
                    (void*)cusz::predictor_quantizer::c_lorenzo_1d1l<Block, Data, Quant>,  //
                    grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);
            if (ndim == 2)  // compile time?
                hipLaunchKernel(
                    (void*)cusz::predictor_quantizer::c_lorenzo_2d1l<Block, Data, Quant>,  //
                    grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);
            else if (ndim == 3)  // compile time?
                hipLaunchKernel(
                    (void*)cusz::predictor_quantizer::c_lorenzo_3d1l<Block, Data, Quant>,  //
                    grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);
#endif
            CHECK_CUDA(hipDeviceSynchronize());
        }

        if (ctx->skip_huff) {
            auto q = mem::CreateHostSpaceAndMemcpyFromDevice(d_q, m->len);
            io::WriteArrayToBinary(fo_q, q, m->len);
            // TODO log
            delete[] q, hipFree(d_q), hipFree(d_data), exit(0);
        }

        // handle outlier
        ::cusz::impl::PruneGatherAsCSR(d_data, MxM, M /*lda*/, M /*m*/, M /*n*/, m->nnz, &fo_outlier);

        // TODO handle metadata
        std::tie(m->total_bits, m->total_uint, m->huff_metadata_size) =
            lossless::interface::HuffmanEncode<Quant, Huff>(fo_q, d_q, m->len, ctx->h_chunksize, m->cap);

        cout << log_info << "Compression finished, saved Huffman encoded quant.code.\n" << endl;
    }

COMPRESS_END:
    hipFree(d_data);
}

template <int ndim, typename Data, int QuantByte, int HuffByte>
void cusz::interface::Decompress2(ctx_t* ctx, typename MetadataTrait<ndim>::metadata_t* m)
{
    typedef typename MetadataTrait<ndim>::metadata_t metadata_t;
    typedef typename QuantTrait<QuantByte>::Quant    Quant;
    typedef typename HuffTrait<HuffByte>::Huff       Huff;

    static const int Block = MetadataTrait<ndim>::Block;

    // todo, put in context
    string fo_x       = ctx->get_fname() + ".szx";
    string fi_qbase   = ctx->get_fname() + ".b" + std::to_string(QuantByte * 8);
    string fi_outlier = fi_qbase + ".outlier";

    auto M   = ::cusz::impl::GetEdgeOfReinterpretedSquare(m->len);
    auto MxM = M * M;

    Quant* xq;
    // step 1: read from filesystem or do Huffman decoding to get quant code
    if (ctx->skip_huff) { xq = io::ReadBinaryToNewArray<Quant>(fi_qbase, m->len); }
    else {
        xq = ::lossless::interface::HuffmanDecode<Quant, Huff>(
            fi_qbase, m->len, ctx->h_chunksize, m->total_uint, m->cap);
        if (ctx->verify_huffman) cusz::impl::VerifyHuffman<ndim, Data, QuantByte>(ctx, m, xq);
    }
    auto d_xq      = mem::CreateDeviceSpaceAndMemcpyFromHost(xq, m->len);
    auto d_outlier = mem::CreateCUDASpace<Data>(MxM);

    cusz::impl::ScatterFromCSR(d_outlier, MxM, M, M, M, &m->nnz, &fi_outlier);

    metadata_t* d_m;
    hipMalloc(&d_m, sizeof(metadata_t));
    hipMemcpy(d_m, m, sizeof(metadata_t), hipMemcpyHostToDevice);

    auto d_xdata = mem::CreateCUDASpace<Data>(m->len);

    {  // Lorenzo
        void* args[] = {&d_m, &d_xdata, &d_xq};
        dim3  grid_dim(m->nb0, m->nb1, m->nb2), block_dim(m->b, m->b, m->b);
        // size_t cache_size = Block;
        // for (auto i = 0; i < ndim - 1; i++) cache_size *= Block;

#if __cplusplus >= 201703L
        hipLaunchKernel(
            (void*)unzip::Lorenzo_nd1l<ndim>::Call<Block, Data, Quant>,  //
            grid_dim, block_dim, args, 0, nullptr);
#elif __cplusplus >= 201402L
        if (ndim == 1)  // compile time?
            hipLaunchKernel(
                (void*)cusz::predictor_quantizer::x_lorenzo_1d1l<Block, Data, Quant>,  //
                grid_dim, block_dim, args, 0, nullptr);
        if (ndim == 2)  // compile time?
            hipLaunchKernel(
                (void*)cusz::predictor_quantizer::x_lorenzo_2d1l<Block, Data, Quant>,  //
                grid_dim, block_dim, args, 0, nullptr);
        else if (ndim == 3)  // compile time?
            hipLaunchKernel(
                (void*)cusz::predictor_quantizer::x_lorenzo_3d1l<Block, Data, Quant>,  //
                grid_dim, block_dim, args, 0, nullptr);
#endif
        CHECK_CUDA(hipDeviceSynchronize());
    }
    auto xdata = mem::CreateHostSpaceAndMemcpyFromDevice(d_xdata, m->len);
    if (not ctx->skip_writex)
        io::WriteArrayToBinary(fo_x, xdata, m->len);
    else
        cout << log_info << "Skipped writing unzipped data to filesystem." << endl;

    delete[] xdata;
    delete[] xq;
    hipFree(d_xdata);
    hipFree(d_outlier);
    hipFree(d_xq);
}

template <int ndim, typename Data, int QuantByte>
void cusz::impl::VerifyHuffman(
    ctx_t*                                    ctx,
    typename MetadataTrait<ndim>::metadata_t* m,
    typename QuantTrait<QuantByte>::Quant*    xq)
{
    typedef typename MetadataTrait<ndim>::metadata_t metadata_t;
    typedef typename QuantTrait<QuantByte>::Quant    Quant;

    static const int Block = MetadataTrait<ndim>::Block;

    // TODO error handling from invalid read
    // cout << log_info << "Redo PdQ just to get quantization dump." << endl;

    auto  data   = io::ReadBinaryToNewArray<Data>(ctx->get_fname(), m->len);
    Data* d_data = mem::CreateDeviceSpaceAndMemcpyFromHost(data, m->len);
    auto  d_q    = mem::CreateCUDASpace<Quant>(m->len);

    metadata_t* d_m;
    hipMalloc(&d_m, sizeof(metadata_t));
    hipMemcpy(d_m, m, sizeof(metadata_t), hipMemcpyHostToDevice);

    {  // Lorenzo
        void*  args[] = {&d_m, &d_data, &d_q};
        dim3   grid_dim(m->nb0, m->nb1, m->nb2), block_dim(m->b, m->b, m->b);
        size_t cache_size = Block;
        for (auto i = 0; i < m->ndim - 1; i++) cache_size *= Block;

#if __cplusplus >= 201703L
        hipLaunchKernel(
            (void*)zip::Lorenzo_nd1l<ndim>::Call<Block, Data, Quant>,  //
            grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);
#elif __cplusplus >= 201402L
        if (ndim == 1)  // compile time?
            hipLaunchKernel(
                (void*)cusz::predictor_quantizer::c_lorenzo_1d1l<Block, Data, Quant>,  //
                grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);
        if (ndim == 2)  // compile time?
            hipLaunchKernel(
                (void*)cusz::predictor_quantizer::c_lorenzo_2d1l<Block, Data, Quant>,  //
                grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);
        else if (ndim == 3)  // compile time?
            hipLaunchKernel(
                (void*)cusz::predictor_quantizer::c_lorenzo_3d1l<Block, Data, Quant>,  //
                grid_dim, block_dim, args, cache_size * sizeof(Data), nullptr);
#endif
        CHECK_CUDA(hipDeviceSynchronize());
    }

    auto len    = m->len;
    auto veri_q = mem::CreateHostSpaceAndMemcpyFromDevice(d_q, len);

    auto count = 0;
    for (auto i = 0; i < len; i++)
        if (xq[i] != veri_q[i]) count++;
    if (count != 0)
        cerr << log_err << "percentage of not being equal: " << count / (1.0 * len) << "\n";
    else
        cout << log_info << "Decoded correctly." << endl;

    if (count != 0) {
        auto n_chunk = (len - 1) / ctx->h_chunksize + 1;
        for (auto c = 0; c < n_chunk; c++) {
            auto chunk_id_printed   = false;
            auto prev_point_printed = false;
            for (auto i = 0; i < ctx->h_chunksize; i++) {
                auto idx = i + c * ctx->h_chunksize;
                if (idx >= len) break;
                if (xq[idx] != xq[idx]) {
                    if (not chunk_id_printed) {
                        cerr << "chunk id: " << c << "\t";
                        cerr << "start@ " << c * ctx->h_chunksize << "\tend@ " << (c + 1) * ctx->h_chunksize - 1
                             << endl;
                        chunk_id_printed = true;
                    }
                    if (not prev_point_printed) {
                        if (idx != c * ctx->h_chunksize) {  // not first point
                            cerr << "PREV-idx:" << idx - 1 << "\t" << xq[idx - 1] << "\t" << xq[idx - 1] << endl;
                        }
                        else {
                            cerr << "wrong at first point!" << endl;
                        }
                        prev_point_printed = true;
                    }
                    cerr << "idx:" << idx << "\tdecoded: " << xq[idx] << "\tori: " << xq[idx] << endl;
                }
            }
        }
    }

    hipFree(d_q), hipFree(d_data);
    delete[] veri_q, delete[] data;
    // end of if count
}
