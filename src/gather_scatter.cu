/**
 * @file gather_scatter.cu
 * @author Jiannan Tian
 * @brief Gather/scatter method to handle cuSZ prediction outlier.
 * @version 0.1
 * @date 2020-09-20
 * Created on 2020-09-10
 *
 * @copyright Copyright (c) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <cassert>
#include <iostream>
#include "argparse.hh"
using std::cout;
using std::endl;

#include "cuda_error_handling.cuh"
#include "format.hh"
#include "gather_scatter.cuh"
#include "io.hh"

using handle_t = hipsparseHandle_t;
using stream_t = hipStream_t;
using descr_t  = hipsparseMatDescr_t;

template <typename DType>
void cusz::impl::GatherAsCSR(DType* d_A, size_t lenA, size_t ldA, size_t m, size_t n, int* nnz, std::string* fo)
{
    uint8_t* outbin;
    size_t   lrp, lci, lv, ltotal;

    {
        handle_t handle        = nullptr;
        stream_t stream        = nullptr;
        descr_t  descr         = nullptr;
        int*     d_nnz_per_row = nullptr;
        int*     d_row_ptr     = nullptr;
        int*     d_col_ind     = nullptr;
        DType*   d_csr_val     = nullptr;

        // clang-format off
        CHECK_CUDA(hipStreamCreateWithFlags   ( &stream, hipStreamNonBlocking        )); // 1. create stream
        CHECK_CUSPARSE(hipsparseCreate          ( &handle                               )); // 2. create handle
        CHECK_CUSPARSE(hipsparseSetStream       (  handle, stream                       )); // 3. bind stream
        CHECK_CUSPARSE(hipsparseCreateMatDescr  ( &descr                                )); // 4. create descr
        CHECK_CUSPARSE(hipsparseSetMatIndexBase (  descr,  HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
        CHECK_CUSPARSE(hipsparseSetMatType      (  descr,  HIPSPARSE_MATRIX_TYPE_GENERAL )); // type

        CHECK_CUDA(hipMalloc((void**)&d_nnz_per_row, sizeof(int) * m));

        CHECK_CUSPARSE(hipsparseSnnz(
            handle, HIPSPARSE_DIRECTION_ROW, // parsed by row
            m, n, descr, d_A, ldA,          // descrption of d_A
            d_nnz_per_row, nnz)             // output
        );

        lrp    = sizeof(int)   * (m + 1);
        lci    = sizeof(int)   * *nnz;
        lv     = sizeof(DType) * *nnz;
        ltotal = lrp + lci + lv;
        outbin = new uint8_t[ltotal];
        CHECK_CUDA(hipMalloc((void**)&d_row_ptr, lrp));
        CHECK_CUDA(hipMalloc((void**)&d_col_ind, lci));
        CHECK_CUDA(hipMalloc((void**)&d_csr_val, lv ));

        CHECK_CUSPARSE(hipsparseSdense2csr(
            handle,                             //
            m, n, descr, d_A, ldA,              // descritpion of d_A
            d_nnz_per_row,                      // prefileld by nnz() func
            d_csr_val, d_row_ptr, d_col_ind)    // output
        );
        CHECK_CUDA(hipDeviceSynchronize());

        CHECK_CUDA(hipMemcpy(outbin,             d_row_ptr, lrp, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(outbin + lrp,       d_col_ind, lci, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(outbin + lrp + lci, d_csr_val, lv,  hipMemcpyDeviceToHost));

        if (d_row_ptr)  hipFree(d_row_ptr);
        if (d_col_ind)  hipFree(d_col_ind);
        if (d_csr_val)  hipFree(d_csr_val);
        if (d_nnz_per_row) hipFree(d_nnz_per_row);
        if (handle)     hipsparseDestroy(handle);
        if (stream)     hipStreamDestroy(stream);
        if (descr)      hipsparseDestroyMatDescr(descr);
        // clang-format on
    }

    // cout << log_dbg << "outlier_bin byte length:\t" << ltotal << endl;
    io::WriteArrayToBinary(*fo, outbin, ltotal);
    delete[] outbin;
};

template void
cusz::impl::GatherAsCSR<float>(float* d_A, size_t lenA, size_t ldA, size_t m, size_t n, int* nnz, std::string* fo);

template <typename DType>
void cusz::impl::ScatterFromCSR(DType* d_A, size_t lenA, size_t ldA, size_t m, size_t n, int* nnz, std::string* fi)
{
    // clang-format off
    auto lrp         = sizeof(int) * (ldA + 1);
    auto lci         = sizeof(int) * *nnz;
    auto lv          = sizeof(DType) * *nnz;
    auto l_total     = lrp + lci + lv;
    auto outlier_bin = io::ReadBinaryFile<uint8_t>(*fi, l_total);
    auto row_ptr     = reinterpret_cast<int*>(outlier_bin);
    auto col_ind     = reinterpret_cast<int*>(outlier_bin + lrp);
    auto csr_val     = reinterpret_cast<DType*>(outlier_bin + lrp + lci);  // TODO template
    // clang-format on

    {
        handle_t handle    = nullptr;
        stream_t stream    = nullptr;
        descr_t  descr     = nullptr;
        int*     d_row_ptr = nullptr;
        int*     d_col_ind = nullptr;
        DType*   d_csr_val = nullptr;

        // clang-format off
        CHECK_CUDA(hipStreamCreateWithFlags   ( &stream, hipStreamNonBlocking        )); // 1. create stream
        CHECK_CUSPARSE(hipsparseCreate          ( &handle                               )); // 2. create handle
        CHECK_CUSPARSE(hipsparseSetStream       (  handle, stream                       )); // 3. bind stream
        CHECK_CUSPARSE(hipsparseCreateMatDescr  ( &descr                                )); // 4. create descr
        CHECK_CUSPARSE(hipsparseSetMatIndexBase (  descr,  HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
        CHECK_CUSPARSE(hipsparseSetMatType      (  descr,  HIPSPARSE_MATRIX_TYPE_GENERAL )); // type

        CHECK_CUDA(hipMalloc( (void**)&d_row_ptr,   lrp ));
        CHECK_CUDA(hipMalloc( (void**)&d_col_ind,   lci ));
        CHECK_CUDA(hipMalloc( (void**)&d_csr_val,      lv    ));
        CHECK_CUDA(hipMemcpy( d_row_ptr, row_ptr, lrp, hipMemcpyHostToDevice ));
        CHECK_CUDA(hipMemcpy( d_col_ind, col_ind, lci, hipMemcpyHostToDevice ));
        CHECK_CUDA(hipMemcpy( d_csr_val, csr_val, lv,  hipMemcpyHostToDevice ));

        CHECK_CUSPARSE(hipsparseScsr2dense(handle, m, n, descr, d_csr_val, d_row_ptr, d_col_ind, d_A, ldA));
        CHECK_CUDA(hipDeviceSynchronize());

        if (d_row_ptr) hipFree(d_row_ptr);
        if (d_col_ind) hipFree(d_col_ind);
        if (d_csr_val) hipFree(d_csr_val);
        if (handle)    hipsparseDestroy(handle);
        if (stream)    hipStreamDestroy(stream);
        if (descr)     hipsparseDestroyMatDescr(descr);
        // clang-format on
    }

    cout << log_info << "Extracted outlier from CSR format." << endl;

    delete[] outlier_bin;
}

template void
cusz::impl::ScatterFromCSR<float>(float* d_A, size_t lenA, size_t ldA, size_t m, size_t n, int* nnz, std::string* fi);

void cusz::impl::PruneGatherAsCSR(
    float*       d_A,  //
    size_t       lenA,
    const int    lda,
    const int    m,
    const int    n,
    int&         nnzC,
    std::string* fo,
    argpack*     ap)
{
    handle_t handle       = nullptr;
    stream_t stream       = nullptr;
    descr_t  descr        = nullptr;
    int*     d_row_ptr    = nullptr;
    int*     d_col_ind    = nullptr;
    float*   d_csr_val    = nullptr;
    size_t   lworkInBytes = 0;
    char*    d_work       = nullptr;
    float    threshold    = 0;

    /*timer*/ ap->cusz_events.push_back(new Event("HOST   CONFIG\tcuSPARSE setup"));
    /*timer*/ ap->cusz_events.back()->Start();
    // clang-format off
    CHECK_CUDA(hipStreamCreateWithFlags   ( &stream, hipStreamNonBlocking        )); // 1. create stream
    CHECK_CUSPARSE(hipsparseCreate          ( &handle                               )); // 2. create handle
    CHECK_CUSPARSE(hipsparseSetStream       (  handle, stream                       )); // 3. bind stream
    CHECK_CUSPARSE(hipsparseCreateMatDescr  ( &descr                                )); // 4. create descr
    CHECK_CUSPARSE(hipsparseSetMatIndexBase (  descr,  HIPSPARSE_INDEX_BASE_ZERO     )); // zero based
    CHECK_CUSPARSE(hipsparseSetMatType      (  descr,  HIPSPARSE_MATRIX_TYPE_GENERAL )); // type
    // clang-format on
    /*timer*/ ap->cusz_events.back()->End();

    CHECK_CUDA(hipMalloc((void**)&d_row_ptr, sizeof(int) * (m + 1)));

    // omit for now
    // /*timer*/ ap->cusz_events.push_back(new Event("cuSPARSE buffer-size-ext"));
    // /*timer*/ ap->cusz_events.back()->Start();
    CHECK_CUSPARSE(hipsparseSpruneDense2csr_bufferSizeExt(  //
        handle, m, n, d_A, lda, &threshold, descr, d_csr_val, d_row_ptr, d_col_ind, &lworkInBytes));
    // /*timer*/ ap->cusz_events.back()->End();

    // printf("lworkInBytes (prune) = %lld \n", (long long)lworkInBytes);
    if (nullptr != d_work) hipFree(d_work);

    CHECK_CUDA(hipMalloc((void**)&d_work, lworkInBytes));

    /*timer*/ ap->cusz_events.push_back(new Event("KERNEL LOSSY\tcuSPARSE compute row_ptr and nnz"));
    /*timer*/ ap->cusz_events.back()->Start();
    /* step 4: compute row_ptrC and nnzC */
    CHECK_CUSPARSE(hipsparseSpruneDense2csrNnz(  //
        handle, m, n, d_A, lda, &threshold, descr, d_row_ptr, &nnzC, d_work));
    CHECK_CUDA(hipDeviceSynchronize());
    /*timer*/ ap->cusz_events.back()->End();

    if (0 == nnzC) cout << log_info << "No outlier." << endl;

    /* step 5: compute col_indC and csr_valC */
    CHECK_CUDA(hipMalloc((void**)&d_col_ind, sizeof(int) * nnzC));
    CHECK_CUDA(hipMalloc((void**)&d_csr_val, sizeof(float) * nnzC));

    /*timer*/ ap->cusz_events.push_back(new Event("KERNEL LOSSY\tcuSPARSE compute col_idx and csr_val"));
    /*timer*/ ap->cusz_events.back()->Start();
    CHECK_CUSPARSE(hipsparseSpruneDense2csr(  //
        handle, m, n, d_A, lda, &threshold, descr, d_csr_val, d_row_ptr, d_col_ind, d_work));
    CHECK_CUDA(hipDeviceSynchronize());
    /*timer*/ ap->cusz_events.back()->End();

    /* step 6: output C */
    auto lrp    = sizeof(int) * (m + 1);
    auto lci    = sizeof(int) * nnzC;
    auto lv     = sizeof(float) * nnzC;
    auto ltotal = lrp + lci + lv;
    auto outbin = new uint8_t[ltotal];

    /*timer*/ ap->cusz_events.push_back(new Event("PCIe   d2h\tmemcpy csr to host"));
    /*timer*/ ap->cusz_events.back()->Start();
    // clang-format off
    CHECK_CUDA(hipMemcpy(outbin,             d_row_ptr, lrp, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(outbin + lrp,       d_col_ind, lci, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(outbin + lrp + lci, d_csr_val, lv,  hipMemcpyDeviceToHost));
    // clang-format on
    /*timer*/ ap->cusz_events.back()->End();

    /*timer*/ ap->cusz_events.push_back(new Event("HOST   I/O\twrite csr to filesystem"));
    /*timer*/ ap->cusz_events.back()->Start();
    io::WriteArrayToBinary(*fo, outbin, ltotal);
    /*timer*/ ap->cusz_events.back()->End();

    if (d_A) hipFree(d_A);
    if (d_row_ptr) hipFree(d_row_ptr);
    if (d_col_ind) hipFree(d_col_ind);
    if (d_csr_val) hipFree(d_csr_val);
    if (handle) hipsparseDestroy(handle);
    if (stream) hipStreamDestroy(stream);
    if (descr) hipsparseDestroyMatDescr(descr);
    if (outbin) delete[] outbin;
}
