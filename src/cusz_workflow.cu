/**
 * @file cusz_workflow.cu
 * @author Jiannan Tian
 * @brief Workflow of cuSZ.
 * @version 0.1
 * @date 2020-09-20
 * Created on: 2020-02-12
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <cxxabi.h>
#include <bitset>
#include <cstdlib>
#include <exception>
#include <iostream>
#include <typeinfo>

// #if __cplusplus >= 201103L

#include <type_traits>
#include "ad_hoc_types.hh"
#include "analysis_utils.hh"
#include "argparse.hh"
#include "autotune.h"
#include "constants.hh"
#include "cuda_error_handling.cuh"
#include "cuda_mem.cuh"
#include "cusz_dryrun.cuh"
#include "cusz_dualquant.cuh"
#include "cusz_workflow.cuh"
#include "filter.cuh"
#include "format.hh"
#include "gather_scatter.cuh"
#include "huffman_workflow.cuh"
#include "io.hh"
#include "timer.hh"
#include "verify.hh"

using std::cerr;
using std::cout;
using std::endl;
using std::string;
using uint8__t = uint8_t;

const int gpu_B_1d = 256;
const int gpu_B_2d = 16;
const int gpu_B_3d = 8;

typedef std::tuple<size_t, size_t, size_t> tuple3ul;

template <typename Data, typename Quant>
void cusz::impl::PdQ(Data* d_d, Quant* d_q, size_t* dims, double* eb_variants)
{
    auto  d_dims        = mem::CreateDeviceSpaceAndMemcpyFromHost(dims, 16);
    auto  d_eb_variants = mem::CreateDeviceSpaceAndMemcpyFromHost(eb_variants, 4);
    void* args[]        = {&d_d, &d_q, &d_dims, &d_eb_variants};

    if (dims[nDIM] == 1) {
        dim3 block_num(dims[nBLK0]);
        dim3 thread_num(gpu_B_1d);
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::c_lorenzo_1d1l<Data, Quant, gpu_B_1d>,  //
            block_num, thread_num, args, 0, nullptr);
    }
    else if (dims[nDIM] == 2) {
        dim3 block_num(dims[nBLK0], dims[nBLK1]);
        dim3 thread_num(gpu_B_2d, gpu_B_2d);
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::c_lorenzo_2d1l<Data, Quant, gpu_B_2d>,  //
            block_num, thread_num, args, (gpu_B_2d + 1) * (gpu_B_2d + 1) * sizeof(Data), nullptr);
    }
    else if (dims[nDIM] == 3) {
        dim3 block_num(dims[nBLK0], dims[nBLK1], dims[nBLK2]);
        dim3 thread_num(gpu_B_3d, gpu_B_3d, gpu_B_3d);
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::c_lorenzo_3d1l<Data, Quant, gpu_B_3d>,  //
            block_num, thread_num, args, (gpu_B_3d + 1) * (gpu_B_3d + 1) * (gpu_B_3d + 1) * sizeof(Data), nullptr);
    }
    HANDLE_ERROR(hipDeviceSynchronize());
}

template <typename Data, typename Quant>
void cusz::impl::ReversedPdQ(Data* d_xd, Quant* d_q, Data* d_outlier, size_t* dims, double _2eb)
{
    auto  d_dims = mem::CreateDeviceSpaceAndMemcpyFromHost(dims, 16);
    void* args[] = {&d_xd, &d_outlier, &d_q, &d_dims, &_2eb};

    if (dims[nDIM] == 1) {
        const static size_t p = gpu_B_1d;

        dim3 thread_num(p);
        dim3 block_num((dims[nBLK0] - 1) / p + 1);
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::x_lorenzo_1d1l<Data, Quant, gpu_B_1d>, block_num, thread_num, args, 0,
            nullptr);
    }
    else if (dims[nDIM] == 2) {
        const static size_t p = gpu_B_2d;

        dim3 thread_num(p, p);
        dim3 block_num(
            (dims[nBLK0] - 1) / p + 1,   //
            (dims[nBLK1] - 1) / p + 1);  //
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::x_lorenzo_2d1l<Data, Quant, gpu_B_2d>, block_num, thread_num, args, 0,
            nullptr);
    }
    else if (dims[nDIM] == 3) {
        const static size_t p = gpu_B_3d;

        dim3 thread_num(p, p, p);
        dim3 block_num(
            (dims[nBLK0] - 1) / p + 1,   //
            (dims[nBLK1] - 1) / p + 1,   //
            (dims[nBLK2] - 1) / p + 1);  //
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::x_lorenzo_3d1l<Data, Quant, gpu_B_3d>, block_num, thread_num, args, 0,
            nullptr);
    }
    else {
        cerr << log_err << "no 4D" << endl;
    }
    hipDeviceSynchronize();

    hipFree(d_dims);
}

template <typename Data, typename Quant>
void cusz::impl::VerifyHuffman(
    string const& fi,
    size_t        len,
    Quant*        xq,
    int           chunk_size,
    size_t*       dims,
    double*       eb_variants)
{
    // TODO error handling from invalid read
    cout << log_info << "Redo PdQ just to get quantization dump." << endl;

    auto  veri_data   = io::ReadBinaryFile<Data>(fi, len);
    Data* veri_d_data = mem::CreateDeviceSpaceAndMemcpyFromHost(veri_data, len);
    auto  veri_d_q    = mem::CreateCUDASpace<Quant>(len);
    PdQ(veri_d_data, veri_d_q, dims, eb_variants);

    auto veri_q = mem::CreateHostSpaceAndMemcpyFromDevice(veri_d_q, len);

    auto count = 0;
    for (auto i = 0; i < len; i++)
        if (xq[i] != veri_q[i]) count++;
    if (count != 0)
        cerr << log_err << "percentage of not being equal: " << count / (1.0 * len) << "\n";
    else
        cout << log_info << "Decoded correctly." << endl;

    if (count != 0) {
        // auto chunk_size = ap->huffman_chunk;
        auto n_chunk = (len - 1) / chunk_size + 1;
        for (auto c = 0; c < n_chunk; c++) {
            auto chunk_id_printed   = false;
            auto prev_point_printed = false;
            for (auto i = 0; i < chunk_size; i++) {
                auto idx = i + c * chunk_size;
                if (idx >= len) break;
                if (xq[idx] != xq[idx]) {
                    if (not chunk_id_printed) {
                        cerr << "chunk id: " << c << "\t";
                        cerr << "start@ " << c * chunk_size << "\tend@ " << (c + 1) * chunk_size - 1 << endl;
                        chunk_id_printed = true;
                    }
                    if (not prev_point_printed) {
                        if (idx != c * chunk_size) {  // not first point
                            cerr << "PREV-idx:" << idx - 1 << "\t" << xq[idx - 1] << "\t" << xq[idx - 1] << endl;
                        }
                        else {
                            cerr << "wrong at first point!" << endl;
                        }
                        prev_point_printed = true;
                    }
                    cerr << "idx:" << idx << "\tdecoded: " << xq[idx] << "\tori: " << xq[idx] << endl;
                }
            }
        }
    }

    hipFree(veri_d_q);
    hipFree(veri_d_data);
    delete[] veri_q;
    delete[] veri_data;
    // end of if count
}

template <typename Data, typename Quant, typename Huff>
void cusz::interface::Compress(
    argpack*                    ap,
    struct AdHocDataPack<Data>* adp,
    size_t*                     dims,
    double*                     eb_variants,
    int&                        nnz_outlier,
    size_t&                     n_bits,
    size_t&                     n_uInt,
    size_t&                     huffman_metadata_size)
{
    // TODO to use a struct
    // TODO already calculated outside in main()
    size_t len = dims[LEN];

    auto data   = adp->data;
    auto d_data = adp->d_data;
    auto m      = adp->m;
    auto mxm    = adp->mxm;

    if (ap->to_dryrun) {
        logall(log_info, "invoke dry-run");
        DryRun(data, d_data, ap->cx_path2file, dims, eb_variants);
        hipHostFree(data);
        hipFree(d_data);
        exit(0);
    }
    logall(log_info, "invoke zipping");

    auto d_q = mem::CreateCUDASpace<Quant>(len);  // quant. code is not needed for dry-run

    // prediction-quantization
    ::cusz::impl::PdQ(d_data, d_q, dims, eb_variants);
    ::cusz::impl::PruneGatherAsCSR(d_data, mxm, m /*lda*/, m /*m*/, m /*n*/, nnz_outlier, &ap->c_fo_outlier);

    auto fmt_nnz = "(" + std::to_string(nnz_outlier / 1.0 / len * 100) + "%)";
    logall(log_info, "nnz/#outlier:", nnz_outlier, fmt_nnz, "saved");
    hipFree(d_data);  // ad-hoc, release memory for large dataset

    Quant* q;
    if (ap->skip_huffman) {
        q = mem::CreateHostSpaceAndMemcpyFromDevice(d_q, len);
        io::WriteArrayToBinary(ap->c_fo_q, q, len);

        logall(log_info, "to store quant.code directly (Huffman enc skipped)");

        return;
    }

    // autotuning Huffman chunksize
    // subject to change, current `8*` is close to but may note deterministically optimal
    if (ap->autotune_huffman_chunk) {  //
        auto optimal_chunksize = 1;
        auto cuda_core_num     = cusz::tune::GetCUDACoreNum();
        auto cuda_thread_num   = 8 * cuda_core_num;  // empirical value

        while (optimal_chunksize * cuda_thread_num < len) optimal_chunksize *= 2;
        ap->huffman_chunk = optimal_chunksize;
    }

    std::tie(n_bits, n_uInt, huffman_metadata_size) =
        lossless::interface::HuffmanEncode<Quant, Huff>(ap->c_huff_base, d_q, len, ap->huffman_chunk, dims[CAP]);

    logall(log_dbg, "to store Huffman encoded quant.code (default)");

    hipFree(d_q);
}

template <typename Data, typename Quant, typename Huff>
void cusz::interface::Decompress(
    argpack* ap,
    size_t*  dims,
    double*  eb_variants,
    int&     nnz_outlier,
    size_t&  total_bits,
    size_t&  total_uInt,
    size_t&  huffman_metadata_size)
{
    auto dict_size = dims[CAP];
    auto len       = dims[LEN];
    auto m         = ::cusz::impl::GetEdgeOfReinterpretedSquare(len);
    auto mxm       = m * m;

    logall(log_info, "invoke unzip");

    Quant* xq;
    // step 1: read from filesystem or do Huffman decoding to get quant code
    if (ap->skip_huffman) {
        logall(log_info, "load quant.code from filesystem");
        xq = io::ReadBinaryFile<Quant>(ap->x_fi_q, len);
    }
    else {
        logall(log_info, "Huffman decode -> quant.code");
        xq = lossless::interface::HuffmanDecode<Quant, Huff>(
            ap->cx_path2file, len, ap->huffman_chunk, total_uInt, dict_size);
        if (ap->verify_huffman) {
            // TODO check in argpack
            if (ap->x_fi_origin == "") {
                cerr << log_err << "use \"--origin /path/to/origin_data\" to specify the original datum." << endl;
                exit(-1);
            }
            cout << log_info << "Verifying Huffman codec..." << endl;
            ::cusz::impl::VerifyHuffman<Data, Quant>(ap->x_fi_origin, len, xq, ap->huffman_chunk, dims, eb_variants);
        }
    }
    auto d_xq = mem::CreateDeviceSpaceAndMemcpyFromHost(xq, len);

    auto d_outlier = mem::CreateCUDASpace<Data>(mxm);
    ::cusz::impl::ScatterFromCSR<Data>(d_outlier, mxm, m /*lda*/, m /*m*/, m /*n*/, &nnz_outlier, &ap->x_fi_outlier);

    // TODO merge d_outlier and d_data
    auto d_xdata = mem::CreateCUDASpace<Data>(len);
    ::cusz::impl::ReversedPdQ(d_xdata, d_xq, d_outlier, dims, eb_variants[EBx2]);
    auto xdata = mem::CreateHostSpaceAndMemcpyFromDevice(d_xdata, len);

    logall(log_info, "reconstruct error-bounded datum");

    size_t archive_size = 0;
    // TODO huffman chunking metadata
    if (not ap->skip_huffman)
        archive_size += total_uInt * sizeof(Huff)  // Huffman coded
                        + huffman_metadata_size;   // chunking metadata and reverse codebook
    else
        archive_size += len * sizeof(Quant);
    archive_size += nnz_outlier * (sizeof(Data) + sizeof(int)) + (m + 1) * sizeof(int);

    // TODO g++ and clang++ use mangled type_id name, add macro
    // https://stackoverflow.com/a/4541470/8740097
    auto demangle = [](const char* name) {
        int   status = -4;
        char* res    = abi::__cxa_demangle(name, nullptr, nullptr, &status);

        const char* const demangled_name = (status == 0) ? res : name;
        string            ret_val(demangled_name);
        free(res);
        return ret_val;
    };

    if (ap->skip_huffman) {
        cout << log_info << "dtype is \""         //
             << demangle(typeid(Data).name())     // demangle
             << "\", and quant. code type is \""  //
             << demangle(typeid(Quant).name())    // demangle
             << "\"; a CR of no greater than "    //
             << (sizeof(Data) / sizeof(Quant)) << " is expected when Huffman codec is skipped." << endl;
    }

    if (ap->pre_binning) cout << log_info << "Because of 2x2->1 binning, extra 4x CR is added." << endl;

    // TODO move CR out of VerifyData
    if (ap->x_fi_origin != "") {
        logall(log_info, "load the original datum for comparison");

        auto odata = io::ReadBinaryFile<Data>(ap->x_fi_origin, len);
        analysis::VerifyData(
            xdata, odata,
            len,              //
            false,            //
            eb_variants[EB],  //
            archive_size,
            ap->pre_binning ? 4 : 1);  // TODO use template rather than 2x2
        delete[] odata;
    }
    logall(log_info, "output:", ap->cx_path2file + ".szx");

    if (!ap->skip_writex)
        io::WriteArrayToBinary(ap->x_fo_xd, xdata, len);
    else {
        logall(log_dbg, "skipped writing unzipped to filesystem");
    }

    // clean up
    delete[] xdata;
    delete[] xq;
    hipFree(d_xdata);
    hipFree(d_outlier);
    hipFree(d_xq);
}

typedef struct AdHocDataPack<float> adp_f32_t;
namespace szin = cusz::interface;

template void szin::Compress<FP4, UI1, UI4>(argpack*, adp_f32_t*, size_t*, FP8*, int&, size_t&, size_t&, size_t&);
template void szin::Compress<FP4, UI1, UI8>(argpack*, adp_f32_t*, size_t*, FP8*, int&, size_t&, size_t&, size_t&);
template void szin::Compress<FP4, UI1, UI8_2>(argpack*, adp_f32_t*, size_t*, FP8*, int&, size_t&, size_t&, size_t&);
template void szin::Compress<FP4, UI2, UI4>(argpack*, adp_f32_t*, size_t*, FP8*, int&, size_t&, size_t&, size_t&);
template void szin::Compress<FP4, UI2, UI8>(argpack*, adp_f32_t*, size_t*, FP8*, int&, size_t&, size_t&, size_t&);
template void szin::Compress<FP4, UI2, UI8_2>(argpack*, adp_f32_t*, size_t*, FP8*, int&, size_t&, size_t&, size_t&);

template void szin::Decompress<FP4, UI1, UI4>(argpack*, size_t*, FP8*, int&, size_t&, size_t&, size_t&);
template void szin::Decompress<FP4, UI1, UI8>(argpack*, size_t*, FP8*, int&, size_t&, size_t&, size_t&);
template void szin::Decompress<FP4, UI1, UI8_2>(argpack*, size_t*, FP8*, int&, size_t&, size_t&, size_t&);
template void szin::Decompress<FP4, UI2, UI4>(argpack*, size_t*, FP8*, int&, size_t&, size_t&, size_t&);
template void szin::Decompress<FP4, UI2, UI8>(argpack*, size_t*, FP8*, int&, size_t&, size_t&, size_t&);
template void szin::Decompress<FP4, UI2, UI8_2>(argpack*, size_t*, FP8*, int&, size_t&, size_t&, size_t&);

// #endif