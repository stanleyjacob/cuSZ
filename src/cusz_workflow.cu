/**
 * @file cusz_workflow.cu
 * @author Jiannan Tian
 * @brief Workflow of cuSZ.
 * @version 0.1
 * @date 2020-09-21
 * Created on: 2020-02-12
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <cxxabi.h>
#include <bitset>
#include <cassert>
#include <cstdarg>
#include <cstdio>
#include <cstdlib>
#include <exception>
#include <iostream>
#include <string>
#include <type_traits>
#include <typeinfo>

#include "argparse.hh"
#include "argparse2_cusz.hh"
#include "constants.hh"
#include "cuda_error_handling.cuh"
#include "cuda_mem.cuh"
#include "cusz_dryrun.cuh"
#include "cusz_dualquant.cuh"
#include "cusz_workflow.cuh"
#include "filter.cuh"
#include "format.hh"
#include "gather_scatter.cuh"
#include "huffman_workflow.cuh"
#include "io.hh"
#include "metadata.hh"
#include "verify.hh"

using std::cerr;
using std::cout;
using std::endl;
using std::string;
using uint8__t = uint8_t;

const int gpu_B_1d = 256;
const int gpu_B_2d = 16;
const int gpu_B_3d = 8;

// moved to const_device.cuh
__constant__ int    symb_dims[16];
__constant__ double symb_ebs[4];

typedef std::tuple<size_t, size_t, size_t> tuple3ul;

/**
 * @deprecated substitute this in 0.1.1 or higher

 */
template <typename Data, typename Quant>
void cusz::impl::PdQ(Data* d_data, Quant* d_q, size_t* dims_L16, double* ebs_L4)
{
    auto  d_dims_L16 = mem::CreateDeviceSpaceAndMemcpyFromHost(dims_L16, 16);
    auto  d_ebs_L4   = mem::CreateDeviceSpaceAndMemcpyFromHost(ebs_L4, 4);
    void* args[]     = {&d_data, &d_q, &d_dims_L16, &d_ebs_L4};

    if (dims_L16[nDIM] == 1) {
        dim3 grid_dim(dims_L16[nBLK0]);
        dim3 block_dim(gpu_B_1d);
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::c_lorenzo_1d1l<Data, Quant, gpu_B_1d>,  //
            grid_dim, block_dim, args, 0, nullptr);
    }
    else if (dims_L16[nDIM] == 2) {
        dim3 grid_dim(dims_L16[nBLK0], dims_L16[nBLK1]);
        dim3 block_dim(gpu_B_2d, gpu_B_2d);
        // old, use physical padding
        // hipLaunchKernel(
        //     (void*)cusz::predictor_quantizer::c_lorenzo_2d1l<Data, Quant, gpu_B_2d>,  //
        //     grid_dim, block_dim, args, (gpu_B_2d + 1) * (gpu_B_2d + 1) * sizeof(Data), nullptr);
        // new, use virtual padding
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::c_lorenzo_2d1l_virtual_padding<Data, Quant, gpu_B_2d>,  //
            grid_dim, block_dim, args, (gpu_B_2d) * (gpu_B_2d) * sizeof(Data), nullptr);
    }
    else if (dims_L16[nDIM] == 3) {
        dim3 grid_dim(dims_L16[nBLK0], dims_L16[nBLK1], dims_L16[nBLK2]);
        dim3 block_dim(gpu_B_3d, gpu_B_3d, gpu_B_3d);
        // old, use physical padding
        // hipLaunchKernel(
        //     (void*)cusz::predictor_quantizer::c_lorenzo_3d1l<Data, Quant, gpu_B_3d>,  //
        //     grid_dim, block_dim, args, (gpu_B_3d + 1) * (gpu_B_3d + 1) * (gpu_B_3d + 1) * sizeof(Data), nullptr);
        // new, use virtual padding
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::c_lorenzo_3d1l_virtual_padding<Data, Quant, gpu_B_3d>,  //
            grid_dim, block_dim, args, (gpu_B_3d) * (gpu_B_3d) * (gpu_B_3d) * sizeof(Data), nullptr);
    }
    HANDLE_ERROR(hipDeviceSynchronize());
}

template void cusz::impl::PdQ<float, uint8__t>(float* d_data, uint8__t* d_q, size_t* dims_L16, double* ebs_L4);
template void cusz::impl::PdQ<float, uint16_t>(float* d_data, uint16_t* d_q, size_t* dims_L16, double* ebs_L4);
template void cusz::impl::PdQ<float, uint32_t>(float* d_data, uint32_t* d_q, size_t* dims_L16, double* ebs_L4);
// template void cusz::impl::PdQ<double, uint8__t>(double* d_data, uint8__t* d_q, size_t* dims_L16, double* ebs_L4);
// template void cusz::impl::PdQ<double, uint16_t>(double* d_data, uint16_t* d_q, size_t* dims_L16, double* ebs_L4);
// template void cusz::impl::PdQ<double, uint32_t>(double* d_data, uint32_t* d_q, size_t* dims_L16, double* ebs_L4);

/**
 * @brief
 * @deprecated substitute this in 0.1.1 or higher
 *
 * @tparam Data
 * @tparam Quant
 * @param d_xdata
 * @param d_q
 * @param d_outlier
 * @param dims_L16
 * @param _2eb
 */
template <typename Data, typename Quant>
void cusz::impl::ReversedPdQ(Data* d_xdata, Quant* d_q, Data* d_outlier, size_t* dims_L16, double _2eb)
{
    auto  d_dims_L16 = mem::CreateDeviceSpaceAndMemcpyFromHost(dims_L16, 16);
    void* args[]     = {&d_xdata, &d_outlier, &d_q, &d_dims_L16, &_2eb};

    if (dims_L16[nDIM] == 1) {
        const static size_t p = gpu_B_1d;

        dim3 thread_num(p);
        dim3 block_num((dims_L16[nBLK0] - 1) / p + 1);
        hipLaunchKernel(                                                             //
            (void*)cusz::predictor_quantizer::x_lorenzo_1d1l<Data, Quant, gpu_B_1d>,  //
            block_num, thread_num, args, 0, nullptr);
    }
    else if (dims_L16[nDIM] == 2) {
        const static size_t p = gpu_B_2d;

        dim3 thread_num(p, p);
        dim3 block_num(((dims_L16[nBLK0] - 1) / p + 1), ((dims_L16[nBLK1] - 1) / p + 1));
        hipLaunchKernel(                                                             //
            (void*)cusz::predictor_quantizer::x_lorenzo_2d1l<Data, Quant, gpu_B_2d>,  //
            block_num, thread_num, args, 0, nullptr);
    }
    else if (dims_L16[nDIM] == 3) {
        const static size_t p = gpu_B_3d;

        dim3 thread_num(p, p, p);
        dim3 block_num(
            ((dims_L16[nBLK0] - 1) / p + 1), ((dims_L16[nBLK1] - 1) / p + 1), ((dims_L16[nBLK2] - 1) / p + 1));
        hipLaunchKernel(                                                             //
            (void*)cusz::predictor_quantizer::x_lorenzo_3d1l<Data, Quant, gpu_B_3d>,  //
            block_num, thread_num, args, 0, nullptr);
    }
    else {
        cerr << log_err << "no 4D" << endl;
    }
    hipDeviceSynchronize();

    hipFree(d_dims_L16);
}

/**
 * @deprecated in 0.1.1
 */
template <typename Data, typename Quant>
void cusz::impl::VerifyHuffman(
    string const& fi,
    size_t        len,
    Quant*        xq,
    int           chunk_size,
    size_t*       dims_L16,
    double*       ebs_L4)
{
    // TODO error handling from invalid read
    cout << log_info << "Redo PdQ just to get quantization dump." << endl;

    auto  veri_data   = io::ReadBinaryFile<Data>(fi, len);
    Data* veri_d_data = mem::CreateDeviceSpaceAndMemcpyFromHost(veri_data, len);
    auto  veri_d_q    = mem::CreateCUDASpace<Quant>(len);
    PdQ(veri_d_data, veri_d_q, dims_L16, ebs_L4);

    auto veri_q = mem::CreateHostSpaceAndMemcpyFromDevice(veri_d_q, len);

    auto count = 0;
    for (auto i = 0; i < len; i++)
        if (xq[i] != veri_q[i]) count++;
    if (count != 0)
        cerr << log_err << "percentage of not being equal: " << count / (1.0 * len) << "\n";
    else
        cout << log_info << "Decoded correctly." << endl;

    if (count != 0) {
        // auto chunk_size = ap->huffman_chunk;
        auto n_chunk = (len - 1) / chunk_size + 1;
        for (auto c = 0; c < n_chunk; c++) {
            auto chunk_id_printed   = false;
            auto prev_point_printed = false;
            for (auto i = 0; i < chunk_size; i++) {
                auto idx = i + c * chunk_size;
                if (idx >= len) break;
                if (xq[idx] != xq[idx]) {
                    if (not chunk_id_printed) {
                        cerr << "chunk id: " << c << "\t";
                        cerr << "start@ " << c * chunk_size << "\tend@ " << (c + 1) * chunk_size - 1 << endl;
                        chunk_id_printed = true;
                    }
                    if (not prev_point_printed) {
                        if (idx != c * chunk_size) {  // not first point
                            cerr << "PREV-idx:" << idx - 1 << "\t" << xq[idx - 1] << "\t" << xq[idx - 1] << endl;
                        }
                        else {
                            cerr << "wrong at first point!" << endl;
                        }
                        prev_point_printed = true;
                    }
                    cerr << "idx:" << idx << "\tdecoded: " << xq[idx] << "\tori: " << xq[idx] << endl;
                }
            }
        }
    }

    hipFree(veri_d_q);
    hipFree(veri_d_data);
    delete[] veri_q;
    delete[] veri_data;
    // end of if count
}

/**
 * @deprecated soon
 */
template <typename Data, typename Quant, typename Huff>
void cusz::interface::Compress(
    argpack* ap,
    size_t*  dims_L16,
    double*  ebs_L4,
    int&     nnz_outlier,
    size_t&  n_bits,
    size_t&  n_uInt,
    size_t&  huffman_metadata_size)
{
    // int bw = sizeof(Q) * 8;
    // string fo_cdata   = ap->opath + "/" + fi + ".sza";
    // string fo_base    = ap->cx_path2file.substr(ap->cx_path2file.rfind("/") + 1);  // token is "scott"
    // string fo_bcode   = ap->opath + fo_base + ".b" + std::to_string(bw);
    // string fo_outlier = ap->opath + fo_base + ".b" + std::to_string(bw) + ".outlier";
    // cout << fo_outlier << endl;

    // TODO to use a struct
    size_t len = dims_L16[LEN];
    auto   m   = cusz::impl::GetEdgeOfReinterpretedSquare(len);  // row-major mxn matrix
    auto   mxm = m * m;

    cout << log_dbg << "original len:\t" << len << " (padding: " << m << ")" << endl;

    auto data = new Data[mxm]();
    io::ReadBinaryFile<Data>(ap->cx_path2file, data, len);
    T* d_data = mem::CreateDeviceSpaceAndMemcpyFromHost(data, mxm);

    if (ap->to_dryrun) {
        cout << "\n" << log_info << "Commencing dry-run..." << endl;
        DryRun(data, d_data, ap->cx_path2file, dims_L16, ebs_L4);
        exit(0);
    }
    cout << "\n" << log_info << "Commencing compression..." << endl;

    auto d_q = mem::CreateCUDASpace<Quant>(len);  // quant. code is not needed for dry-run

    // prediction-quantization
    ::cusz::impl::PdQ(d_data, d_q, dims_L16, ebs_L4);
    ::cusz::impl::PruneGatherAsCSR(d_data, mxm, m /*lda*/, m /*m*/, m /*n*/, nnz_outlier, &ap->c_fo_outlier);
    cout << log_info << "nnz.outlier:\t" << nnz_outlier << "\t(" << (nnz_outlier / 1.0 / len * 100) << "%)" << endl;

    Quant* q;
    if (ap->skip_huffman) {
        q = mem::CreateHostSpaceAndMemcpyFromDevice(d_q, len);
        io::WriteArrayToBinary(ap->c_fo_q, q, len);
        cout << log_info << "Compression finished, saved quant.code (Huffman skipped).\n" << endl;
        return;
    }

    std::tie(n_bits, n_uInt, huffman_metadata_size) =
        lossless::interface::HuffmanEncode<Quant, Huff>(ap->c_huff_base, d_q, len, ap->huffman_chunk, dims_L16[CAP]);

    cout << log_info << "Compression finished, saved Huffman encoded quant.code.\n";

    delete[] data;
    hipFree(d_data);
    hipFree(d_bcode);
}

/**
 * @deprecated soon
 */
template <typename Data, typename Quant, typename Huff>
void cusz::interface::Decompress(
    argpack* ap,
    size_t*  dims_L16,
    double*  ebs_L4,
    int&     nnz_outlier,
    size_t&  total_bits,
    size_t&  total_uInt,
    size_t&  huffman_metadata_size)
{
    auto dict_size = dims_L16[CAP];
    auto len       = dims_L16[LEN];
    auto m         = ::cusz::impl::GetEdgeOfReinterpretedSquare(len);
    auto mxm       = m * m;

    cout << log_info << "Commencing decompression..." << endl;

    Quant* xq;
    // step 1: read from filesystem or do Huffman decoding to get quant code
    if (ap->skip_huffman) {
        cout << log_info << "Getting quant.code from filesystem... (Huffman encoding was skipped.)" << endl;
        xq = io::ReadBinaryFile<Quant>(ap->x_fi_q, len);
    }
    else {
        cout << log_info << "Huffman decoding into quant.code." << endl;
        xq = lossless::interface::HuffmanDecode<Quant, Huff>(
            ap->cx_path2file, len, ap->huffman_chunk, total_uInt, dict_size);
        if (ap->do_verify_huffman) {
            // TODO check in argpack
            if (ap->x_fi_origin == "") {
                cerr << log_err << "use \"--orogin /path/to/origin_data\" to specify the original dataum." << endl;
                exit(-1);
            }
            cout << log_info << "Verifying Huffman codec..." << endl;
            ::cusz::impl::VerifyHuffman<Data, Quant>(ap->x_fi_origin, len, xq, ap->huffman_chunk, dims_L16, ebs_L4);
        }
    }
    auto d_q = mem::CreateDeviceSpaceAndMemcpyFromHost(xq, len);

    auto d_outlier = mem::CreateCUDASpace<Data>(mxm);
    ::cusz::impl::ScatterFromCSR<Data>(d_outlier, mxm, m /*lda*/, m /*m*/, m /*n*/, &nnz_outlier, &ap->x_fi_outlier);

    // TODO merge d_outlier and d_data
    auto d_xdata = mem::CreateCUDASpace<Data>(len);
    ::cusz::impl::ReversedPdQ(d_xdata, d_q, d_outlier, dims_L16, ebs_L4[EBx2]);
    auto xdata = mem::CreateHostSpaceAndMemcpyFromDevice(d_xdata, len);

    cout << log_info << "Decompression finished.\n\n";

    size_t archive_size = 0;
    // TODO huffman chunking metadata
    if (not ap->skip_huffman)
        archive_size += total_uInt * sizeof(Huff)  // Huffman coded
                        + huffman_metadata_size;   // chunking metadata and reverse codebook
    else
        archive_size += len * sizeof(Quant);
    archive_size += nnz_outlier * (sizeof(Data) + sizeof(int)) + (m + 1) * sizeof(int);

    // TODO g++ and clang++ use mangled type_id name, add macro
    // https://stackoverflow.com/a/4541470/8740097
    auto demangle = [](const char* name) {
        int   status = -4;
        char* res    = abi::__cxa_demangle(name, nullptr, nullptr, &status);

        const char* const demangled_name = (status == 0) ? res : name;
        string            ret_val(demangled_name);
        free(res);
        return ret_val;
    };

    if (ap->skip_huffman) {
        cout << log_info << "Data is \""          //
             << demangle(typeid(Data).name())     // demangle
             << "\", and quant. code type is \""  //
             << demangle(typeid(Quant).name())    // demangle
             << "\"; a CR of no greater than "    //
             << (sizeof(Data) / sizeof(Quant)) << " is expected when Huffman codec is skipped." << endl;
    }

    if (ap->pre_binning) cout << log_info << "Because of 2x2->1 binning, extra 4x CR is added." << endl;
    if (not ap->skip_huffman) {
        cout << log_info
             << "Huffman metadata of chunking and reverse codebook size (in bytes): " << huffman_metadata_size << endl;
        cout << log_info << "Huffman coded output size: " << total_uInt * sizeof(Huff) << endl;
    }

    // TODO move CR out of VerifyData
    if (ap->x_fi_origin != "") {
        cout << log_info << "To compare with the original datum" << endl;
        auto odata = io::ReadBinaryFile<T>(ap->x_fi_origin, len);
        analysis::VerifyData(
            xdata, odata,
            len,         //
            false,       //
            ebs_L4[EB],  //
            archive_size,
            ap->pre_binning ? 4 : 1);  // TODO use template rather than 2x2
        cout << log_info << "Decompressed file is written to " << ap->cx_path2file << ".szx." << endl;
        cout << log_info << "Please use compressed data (*.sz) to calculate final comp ratio (w/ gzip)." << endl;
        delete[] odata;
    }

    if (!ap->skip_writex)
        io::WriteArrayToBinary(ap->x_fo_xd, xdata, len);
    else {
        cout << log_info << "Skipped writing unzipped to filesystem." << endl;
    }

    // clean up
    delete[] xdata;
    delete[] xq;
    hipFree(d_xdata);
    hipFree(d_outlier);
    hipFree(d_q);
}

template void
cusz::interface::Compress<float, uint8__t, uint32_t>(argpack*, size_t*, double*, int&, size_t&, size_t&, size_t&);
template void
cusz::interface::Compress<float, uint8__t, uint64_t>(argpack*, size_t*, double*, int&, size_t&, size_t&, size_t&);
template void
cusz::interface::Compress<float, uint16_t, uint32_t>(argpack*, size_t*, double*, int&, size_t&, size_t&, size_t&);
template void
cusz::interface::Compress<float, uint16_t, uint64_t>(argpack*, size_t*, double*, int&, size_t&, size_t&, size_t&);

template void
cusz::interface::Decompress<float, uint8__t, uint32_t>(argpack*, size_t*, double*, int&, size_t&, size_t&, size_t&);
template void
cusz::interface::Decompress<float, uint8__t, uint64_t>(argpack*, size_t*, double*, int&, size_t&, size_t&, size_t&);
template void
cusz::interface::Decompress<float, uint16_t, uint32_t>(argpack*, size_t*, double*, int&, size_t&, size_t&, size_t&);
template void
cusz::interface::Decompress<float, uint16_t, uint64_t>(argpack*, size_t*, double*, int&, size_t&, size_t&, size_t&);
