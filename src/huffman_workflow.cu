#include "hip/hip_runtime.h"
/**
 * @file huffman_workflow.cu
 * @author Jiannan Tian, Cody Rivera (cjrivera1@crimson.ua.edu)
 * @brief Workflow of Huffman coding.
 * @version 0.1
 * @date 2020-09-20
 * Created on 2020-04-24
 *
 * @copyright Copyright (c) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>

#include <sys/stat.h>
#include <unistd.h>
#include <bitset>
#include <cassert>
#include <cmath>
#include <functional>
#include <iostream>
#include <numeric>
#include <stdexcept>
#include <string>
#include <tuple>
#include <type_traits>
#include <vector>

#include "argparse.hh"
#include "canonical.cuh"
#include "cuda_error_handling.cuh"
#include "cuda_mem.cuh"
#include "dbg_gpu_printing.cuh"
#include "format.hh"
#include "histogram.cuh"
#include "huffman.cuh"
#include "huffman_codec.cuh"
#include "huffman_workflow.cuh"
#include "par_huffman.cuh"
#include "timer.cuh"
#include "types.hh"

int ht_state_num;
int ht_all_nodes;
using uint8__t = uint8_t;

template <typename Q>
void wrapper::GetFrequency(Q* d_bcode, size_t len, unsigned int* d_freq, int dict_size, argpack* ap)
{
    // Parameters for thread and block count optimization

    // Initialize to device-specific values
    int deviceId;
    int maxbytes;
    int numSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&maxbytes, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, deviceId);
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    // Optimize launch
    int numBuckets     = dict_size;
    int numValues      = len;
    int itemsPerThread = 1;
    int RPerBlock      = (maxbytes / (int)sizeof(int)) / (numBuckets + 1);
    int numBlocks      = numSMs;
    hipFuncSetAttribute(reinterpret_cast<const void*>(p2013Histogram<Q), unsigned int>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    // fits to size
    int threadsPerBlock = ((((numValues / (numBlocks * itemsPerThread)) + 1) / 64) + 1) * 64;
    while (threadsPerBlock > 1024) {
        if (RPerBlock <= 1) { threadsPerBlock = 1024; }
        else {
            RPerBlock /= 2;
            numBlocks *= 2;
            threadsPerBlock = ((((numValues / (numBlocks * itemsPerThread)) + 1) / 64) + 1) * 64;
        }
    }

    DeviceEvent* de;
    /*timer*/ de = new DeviceEvent("KERNEL LOSSLESS\thistogramming (end-to-end)");
    p2013Histogram                                                                      //
        <<<numBlocks, threadsPerBlock, ((numBuckets + 1) * RPerBlock) * sizeof(int)>>>  //
        (d_bcode, d_freq, numValues, numBuckets, RPerBlock);
    hipDeviceSynchronize();
    /*timer*/ ap->cusz_events_ms.push_back({de->event_name, de->End()});
    /*timer*/ delete de;

    // TODO make entropy optional
    {
        auto   freq    = mem::CreateHostSpaceAndMemcpyFromDevice(d_freq, dict_size);
        double entropy = 0.0;
        for (auto i = 0; i < dict_size; i++)
            if (freq[i]) {
                auto possibility = freq[i] / (1.0 * len);
                entropy -= possibility * log(possibility);
            }
        cout << log_info << "entropy:\t\t" << entropy << endl;
        delete[] freq;
    }

#ifdef DEBUG_PRINT
    print_histogram<unsigned int><<<1, 32>>>(d_freq, dict_size, dict_size / 2);
    hipDeviceSynchronize();
#endif
}

template <typename H>
void PrintChunkHuffmanCoding(
    size_t* dH_bit_meta,  //
    size_t* dH_uInt_meta,
    size_t  len,
    int     chunk_size,
    size_t  total_bits,
    size_t  total_uInts)
{
    cout << "\n" << log_dbg << "Huffman coding detail start ------" << endl;
    printf("| %s\t%s\t%s\t%s\t%9s\n", "chunk", "bits", "bytes", "uInt", "chunkCR");
    for (size_t i = 0; i < 8; i++) {
        size_t n_byte   = (dH_bit_meta[i] - 1) / 8 + 1;
        auto   chunk_CR = ((double)chunk_size * sizeof(float) / (1.0 * (double)dH_uInt_meta[i] * sizeof(H)));
        printf("| %lu\t%lu\t%lu\t%lu\t%9.6lf\n", i, dH_bit_meta[i], n_byte, dH_uInt_meta[i], chunk_CR);
    }
    cout << "| ..." << endl
         << "| Huff.total.bits:\t" << total_bits << endl
         << "| Huff.total.bytes:\t" << total_uInts * sizeof(H) << endl
         << "| Huff.CR (uInt):\t" << (double)len * sizeof(float) / (total_uInts * 1.0 * sizeof(H)) << endl;
    cout << log_dbg << "coding detail end ----------------" << endl;
    cout << endl;
}

template <typename Q, typename H, typename DATA>
std::tuple<size_t, size_t, size_t> HuffmanEncode(argpack* ap, Q* d_in, size_t len, int chunk_size, int dict_size)
{
    auto f_in = ap->c_huff_base;
    // histogram
    ht_state_num = 2 * dict_size;
    ht_all_nodes = 2 * ht_state_num;
    auto d_freq  = mem::CreateCUDASpace<unsigned int>(ht_all_nodes);

    DeviceEvent* de;

    wrapper::GetFrequency(d_in, len, d_freq, dict_size, ap);

    // Allocate cb memory
    auto d_canonical_cb = mem::CreateCUDASpace<H>(dict_size, 0xff);
    // canonical Huffman; follows H to decide first and entry type
    auto type_bw = sizeof(H) * 8;
    // first, entry, reversed codebook
    // CHANGED first and entry to H type
    auto decode_meta_size = sizeof(H) * (2 * type_bw) + sizeof(Q) * dict_size;
    auto d_decode_meta    = mem::CreateCUDASpace<uint8_t>(decode_meta_size);

    // Get codebooks
    /*timer*/ de = new DeviceEvent("KERNEL LOSSLESS\tparallel-get codebook");
    ParGetCodebook<Q, H>(dict_size, d_freq, d_canonical_cb, d_decode_meta);
    hipDeviceSynchronize();
    /*timer*/ ap->cusz_events_ms.push_back({de->event_name, de->End()});
    /*timer*/ delete de;

    auto decode_meta = mem::CreateHostSpaceAndMemcpyFromDevice(d_decode_meta, decode_meta_size);

    // Non-deflated output
    auto d_h = mem::CreateCUDASpace<H>(len);

    // --------------------------------
    // this is for internal evaluation, not in sz archive
    // auto cb_dump = mem::CreateHostSpaceAndMemcpyFromDevice(d_canonical_cb, dict_size);
    // io::WriteBinaryFile(cb_dump, dict_size, new string(f_in + ".canonized"));
    // --------------------------------

    /*timer*/ de = new DeviceEvent("KERNEL LOSSLESS\tquery space for Hwffman-deflate");
    // fix-length space
    {
        auto blockDim = tBLK_ENCODE;
        auto gridDim  = (len - 1) / blockDim + 1;
        EncodeFixedLen<Q, H><<<gridDim, blockDim>>>(d_in, d_h, len, d_canonical_cb);
        hipDeviceSynchronize();
    }
    /*timer*/ ap->cusz_events_ms.push_back({de->event_name, de->End()});
    /*timer*/ delete de;

    // deflate
    auto n_chunk       = (len - 1) / chunk_size + 1;  // |
    auto d_h_bitwidths = mem::CreateCUDASpace<size_t>(n_chunk);
    // cout << log_dbg << "chunk.size:\t" << chunk_size << endl;
    // cout << log_dbg << "chunk.num:\t" << n_chunk << endl;

    /*timer*/ de = new DeviceEvent("KERNEL LOSSLESS\tHuffman-deflate");
    {
        auto blockDim = tBLK_DEFLATE;
        auto gridDim  = (n_chunk - 1) / blockDim + 1;
        Deflate<H><<<gridDim, blockDim>>>(d_h, len, d_h_bitwidths, chunk_size);
        hipDeviceSynchronize();
    }
    /*timer*/ ap->cusz_events_ms.push_back({de->event_name, de->End()});
    /*timer*/ delete de;

    // dump TODO change to int
    auto h_meta        = new size_t[n_chunk * 3]();
    auto dH_uInt_meta  = h_meta;
    auto dH_bit_meta   = h_meta + n_chunk;
    auto dH_uInt_entry = h_meta + n_chunk * 2;
    // copy back densely Huffman code (dHcode)
    hipMemcpy(dH_bit_meta, d_h_bitwidths, n_chunk * sizeof(size_t), hipMemcpyDeviceToHost);
    // transform in uInt
    memcpy(dH_uInt_meta, dH_bit_meta, n_chunk * sizeof(size_t));
    for_each(dH_uInt_meta, dH_uInt_meta + n_chunk, [&](size_t& i) { i = (i - 1) / (sizeof(H) * 8) + 1; });
    // make it entries
    memcpy(dH_uInt_entry + 1, dH_uInt_meta, (n_chunk - 1) * sizeof(size_t));
    for (auto i = 1; i < n_chunk; i++) dH_uInt_entry[i] += dH_uInt_entry[i - 1];

    // sum bits from each chunk
    auto total_bits  = std::accumulate(dH_bit_meta, dH_bit_meta + n_chunk, (size_t)0);
    auto total_uInts = std::accumulate(dH_uInt_meta, dH_uInt_meta + n_chunk, (size_t)0);

    cout << log_info;
    printf(
        "Huffman bitstream: %lu chunks of size = %d, in %lu uint%lus or %lu bits\n", n_chunk, chunk_size, total_uInts,
        sizeof(H) * 8, total_bits);

    // print densely metadata
    // PrintChunkHuffmanCoding<H>(dH_bit_meta, dH_uInt_meta, len, chunk_size, total_bits, total_uInts);

    /*timer*/ de = new DeviceEvent("PCIe   d2h\tnaive gather Huffman-bitstream");
    // copy back densely Huffman code in units of uInt (regarding endianness)
    // TODO reinterpret_cast
    auto h = new H[total_uInts]();
    for (auto i = 0; i < n_chunk; i++) {
        hipMemcpy(
            h + dH_uInt_entry[i],         // dst
            d_h + i * chunk_size,         // src
            dH_uInt_meta[i] * sizeof(H),  // len in H-uint
            hipMemcpyDeviceToHost);
    }
    /*timer*/ ap->cusz_events_ms.push_back({de->event_name, de->End()});
    /*timer*/ delete de;

    /*timer*/ de = new DeviceEvent("HOST   I/O\twrite Huffman bitstream and metadata");
    // dump bit_meta and uInt_meta
    io::WriteArrayToBinary(f_in + ".hmeta", h_meta + n_chunk, (2 * n_chunk));
    // write densely Huffman code and its metadata
    io::WriteArrayToBinary(f_in + ".hbyte", h, total_uInts);
    // to save first, entry and keys
    io::WriteArrayToBinary(
        f_in + ".canon",                                   //
        reinterpret_cast<uint8_t*>(decode_meta),           //
        sizeof(H) * (2 * type_bw) + sizeof(Q) * dict_size  // first, entry, reversed dict (keys)
    );
    /*timer*/ ap->cusz_events_ms.push_back({de->event_name, de->End()});
    /*timer*/ delete de;

    size_t metadata_size = (2 * n_chunk) * sizeof(decltype(h_meta))              //
                           + sizeof(H) * (2 * type_bw) + sizeof(Q) * dict_size;  // uint8_t

    //////// clean up
    hipFree(d_in);
    hipFree(d_freq);
    hipFree(d_canonical_cb);
    hipFree(d_decode_meta);
    hipFree(d_h);
    hipFree(d_h_bitwidths);
    delete[] h;
    delete[] h_meta;
    delete[] decode_meta;

    return std::make_tuple(total_bits, total_uInts, metadata_size);
}

template <typename Q, typename H, typename DATA>
Q* HuffmanDecode(
    // std::string& f_bcode_base,  //
    argpack* ap,
    size_t   len,
    int      chunk_size,
    int      total_uInts,
    int      dict_size)
{
    auto f_bcode_base = ap->cx_path2file;

    auto type_bw             = sizeof(H) * 8;
    auto canonical_meta      = sizeof(H) * (2 * type_bw) + sizeof(Q) * dict_size;
    auto canonical_singleton = io::ReadBinaryFile<uint8_t>(f_bcode_base + ".canon", canonical_meta);
    hipDeviceSynchronize();

    auto n_chunk  = (len - 1) / chunk_size + 1;
    auto hcode    = io::ReadBinaryFile<H>(f_bcode_base + ".hbyte", total_uInts);
    auto dH_meta  = io::ReadBinaryFile<size_t>(f_bcode_base + ".hmeta", 2 * n_chunk);
    auto blockDim = tBLK_DEFLATE;  // the same as deflating
    auto gridDim  = (n_chunk - 1) / blockDim + 1;

    auto d_xbcode              = mem::CreateCUDASpace<Q>(len);
    auto d_dHcode              = mem::CreateDeviceSpaceAndMemcpyFromHost(hcode, total_uInts);
    auto d_hcode_meta          = mem::CreateDeviceSpaceAndMemcpyFromHost(dH_meta, 2 * n_chunk);
    auto d_canonical_singleton = mem::CreateDeviceSpaceAndMemcpyFromHost(canonical_singleton, canonical_meta);
    hipDeviceSynchronize();

    /*timer*/ auto de = new DeviceEvent("KERNEL LOSSLESS\tHuffman decode");
    Decode<<<gridDim, blockDim, canonical_meta>>>(  //
        d_dHcode, d_hcode_meta, d_xbcode, len, chunk_size, n_chunk, d_canonical_singleton, (size_t)canonical_meta);
    hipDeviceSynchronize();
    /*timer*/ ap->cusz_events_ms.push_back({de->event_name, de->End()});
    /*timer*/ delete de;
    ;

    auto xbcode = mem::CreateHostSpaceAndMemcpyFromDevice(d_xbcode, len);
    hipFree(d_xbcode);
    hipFree(d_dHcode);
    hipFree(d_hcode_meta);
    hipFree(d_canonical_singleton);
    delete[] hcode;
    delete[] dH_meta;
    delete[] canonical_singleton;

    return xbcode;
}

template void wrapper::GetFrequency<uint8__t>(uint8__t*, size_t, unsigned int*, int, argpack*);
template void wrapper::GetFrequency<uint16_t>(uint16_t*, size_t, unsigned int*, int, argpack*);
template void wrapper::GetFrequency<uint32_t>(uint32_t*, size_t, unsigned int*, int, argpack*);

template void PrintChunkHuffmanCoding<uint32_t>(size_t*, size_t*, size_t, int, size_t, size_t);
template void PrintChunkHuffmanCoding<uint64_t>(size_t*, size_t*, size_t, int, size_t, size_t);

template tuple3ul HuffmanEncode<uint8__t, uint32_t, float>(argpack*, uint8__t*, size_t, int, int);
template tuple3ul HuffmanEncode<uint16_t, uint32_t, float>(argpack*, uint16_t*, size_t, int, int);
template tuple3ul HuffmanEncode<uint32_t, uint32_t, float>(argpack*, uint32_t*, size_t, int, int);
template tuple3ul HuffmanEncode<uint8__t, uint64_t, float>(argpack*, uint8__t*, size_t, int, int);
template tuple3ul HuffmanEncode<uint16_t, uint64_t, float>(argpack*, uint16_t*, size_t, int, int);
template tuple3ul HuffmanEncode<uint32_t, uint64_t, float>(argpack*, uint32_t*, size_t, int, int);

template uint8__t* HuffmanDecode<uint8__t, uint32_t, float>(argpack*, size_t, int, int, int);
template uint16_t* HuffmanDecode<uint16_t, uint32_t, float>(argpack*, size_t, int, int, int);
template uint32_t* HuffmanDecode<uint32_t, uint32_t, float>(argpack*, size_t, int, int, int);
template uint8__t* HuffmanDecode<uint8__t, uint64_t, float>(argpack*, size_t, int, int, int);
template uint16_t* HuffmanDecode<uint16_t, uint64_t, float>(argpack*, size_t, int, int, int);
template uint32_t* HuffmanDecode<uint32_t, uint64_t, float>(argpack*, size_t, int, int, int);
// clang-format off
