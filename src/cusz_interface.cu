/**
 * @file cusz_workflow.cu
 * @author Jiannan Tian
 * @brief Workflow of cuSZ.
 * @version 0.2
 * @date 2021-01-16
 * (create) 2020-02-12; (release) 2020-09-20; (rev1) 2021-01-16
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <cxxabi.h>
#include <bitset>
#include <cstdlib>
#include <exception>
#include <iostream>
#include <typeinfo>

// #if __cplusplus >= 201103L

#include <type_traits>

//#include "analysis_utils.hh"
#include "argparse.hh"
#include "autotune.hh"
#include "cusz_interface.cuh"
#include "dryrun.cuh"
#include "dualquant.cuh"
#include "gather_scatter.cuh"
#include "huff_interface.cuh"
#include "lorenzo_trait.cuh"
#include "metadata.hh"
#include "type_trait.hh"
#include "utils/cuda_err.cuh"
#include "utils/cuda_mem.cuh"
#include "utils/format.hh"
#include "utils/io.hh"
#include "utils/verify.hh"

using std::cerr;
using std::cout;
using std::endl;
using std::string;

using namespace cusz::predictor_quantizer::v2;
using namespace cusz::predictor_quantizer::v3;

/*
template <typename Data, typename Quant>
void cusz::impl::VerifyHuffman(
    string const& fi,
    size_t        len,
    Quant*        xq,
    int           chunk_size,
    size_t*       dims,
    double*       eb_variants)
{
    LogAll(log_info, "Redo PdQ just to get quant data.");

    auto  veri_data   = io::ReadBinaryToNewArray<Data>(fi, len);
    Data* veri_d_data = mem::CreateDeviceSpaceAndMemcpyFromHost(veri_data, len);
    auto  veri_d_q    = mem::CreateCUDASpace<Quant>(len);

    PdQ(veri_d_data, veri_d_q, dims, eb_variants);

    auto veri_q = mem::CreateHostSpaceAndMemcpyFromDevice(veri_d_q, len);

    auto count = 0;
    for (auto i = 0; i < len; i++)
        if (xq[i] != veri_q[i]) count++;
    if (count != 0)
        LogAll(log_err, "percentage of not being equal:", count / (1.0 * len));
    else
        LogAll(log_info, "Decoded correctly.");

    if (count != 0) {
        auto n_chunk = (len - 1) / chunk_size + 1;
        for (auto c = 0; c < n_chunk; c++) {
            auto chunk_id_printed = false, prev_point_printed = false;
            for (auto i = 0; i < chunk_size; i++) {
                auto idx = i + c * chunk_size;
                if (idx >= len) break;
                if (xq[idx] != xq[idx]) {
                    if (not chunk_id_printed) {
                        cerr << "chunk id: " << c << "\t"
                             << "start@ " << c * chunk_size << "\tend@ " << (c + 1) * chunk_size - 1 << endl;
                        chunk_id_printed = true;
                    }
                    if (not prev_point_printed) {
                        if (idx != c * chunk_size)  // not first point
                            cerr << "PREV-idx:" << idx - 1 << "\t" << xq[idx - 1] << "\t" << xq[idx - 1] << endl;
                        else
                            cerr << "wrong at first point!" << endl;
                        prev_point_printed = true;
                    }
                    cerr << "idx:" << idx << "\tdecoded: " << xq[idx] << "\tori: " << xq[idx] << endl;
                }
            }
        }
    }

    hipFree(veri_d_q);
    hipFree(veri_d_data);
    delete[] veri_q;
    delete[] veri_data;
}
 */

template <typename T>
auto CopyToBuffer_3D(
    T* __restrict buffer_dst,
    T* __restrict origin_src,
    size_t          portal,
    Index<3>::idx_t part_dims,
    Index<3>::idx_t block_stride,
    Index<3>::idx_t global_stride)
{
    for (auto k = 0; k < part_dims._2; k++)
        for (auto j = 0; j < part_dims._1; j++)
            for (auto i = 0; i < part_dims._0; i++)
                buffer_dst[i + j * block_stride._1 + k * block_stride._2] =
                    origin_src[portal + (i + j * global_stride._1 + k * global_stride._2)];
}

template <typename T, int N = 3>
auto PrintBuffer(T* data, size_t start, Integer3 strides)
{
    cout << "printing buffer\n";
    for (auto k = 0; k < N; k++) {
        for (auto j = 0; j < N; j++) {
            for (auto i = 0; i < N; i++) {  //
                cout << data[start + (i + j * strides._1 + k * strides._2)] << " ";
            }
            cout << "\n";
        }
    }
    cout << endl;
};

// clang-format off
template <bool If_FP, int DataByte, int QuantByte, int HuffByte>
void cusz::interface::Compress(
    argpack* ap,
    struct DataPack<typename DataTrait<If_FP, DataByte>::Data>* adp,
    int&     nnz_outlier,
    size_t&  n_bits,
    size_t&  n_uInt,
    size_t&  huffman_metadata_size,
    bool&    nvcomp_in_use)
{
    // clang-format on
    using Data  = typename DataTrait<If_FP, DataByte>::Data;
    using Quant = typename QuantTrait<QuantByte>::Quant;
    using Huff  = typename HuffTrait<HuffByte>::Huff;

    size_t len = ap->len;

    auto data   = adp->data;
    auto d_data = adp->d_data;
    auto m      = adp->m;
    auto mxm    = adp->mxm;

    if (ap->to_dryrun) {
        LogAll(log_info, "dry-run temporarily not working");
        //        LogAll(log_info, "invoke dry-run");
        //        DryRun(ap, data, d_data, ap->cx_path2file, dims, eb_variants);
        hipHostFree(data);
        hipFree(d_data);
        exit(0);
    }
    LogAll(log_info, "invoke zipping");

    auto d_q = mem::CreateCUDASpace<Quant>(len);  // quant. code is not needed for dry-run

    // prediction-quantization
    {
        if (ap->ndim == 1) {
            LorenzoNdConfig<1, Data, workflow::zip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            c_lorenzo_1d1l<Data, Quant><<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.z_ctx, d_data, d_q);
        }
        if (ap->ndim == 2) {
            LorenzoNdConfig<2, Data, workflow::zip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            c_lorenzo_2d1l<Data, Quant><<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.z_ctx, d_data, d_q);
        }
        else if (ap->ndim == 3) {
            LorenzoNdConfig<3, Data, workflow::zip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            c_lorenzo_3d1l<Data, Quant><<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.z_ctx, d_data, d_q);
        }
        HANDLE_ERROR(hipDeviceSynchronize());
    }

    ::cusz::impl::PruneGatherAsCSR(d_data, mxm, m /*lda*/, m /*m*/, m /*n*/, nnz_outlier, &ap->c_fo_outlier);

    auto fmt_nnz = "(" + std::to_string(nnz_outlier / 1.0 / len * 100) + "%)";
    LogAll(log_info, "nnz/#outlier:", nnz_outlier, fmt_nnz, "saved");
    hipFree(d_data);  // ad-hoc, release memory for large dataset

    Quant* q;
    if (ap->skip_huffman) {
        q = mem::CreateHostSpaceAndMemcpyFromDevice(d_q, len);
        io::WriteArrayToBinary(ap->c_fo_q, q, len);

        LogAll(log_info, "to store quant.code directly (Huffman enc skipped)");

        return;
    }

    // autotuning Huffman chunksize
    // subject to change, current `8*` is close to but may note deterministically optimal
    if (ap->autotune_huffman_chunk) {  //
        auto optimal_chunksize = 1;
        auto cuda_core_num     = cusz::tune::GetCUDACoreNum();
        auto cuda_thread_num   = 8 * cuda_core_num;  // empirical value

        while (optimal_chunksize * cuda_thread_num < len) optimal_chunksize *= 2;
        ap->huffman_chunk = optimal_chunksize;
    }

    if (ap->conduct_partition_experiment) {
        // 3D only
        auto part0     = ap->p0;
        auto part1     = ap->p1;
        auto part2     = ap->p2;
        auto num_part0 = (ap->dim4._0 - 1) / part0 + 1;
        auto num_part1 = (ap->dim4._1 - 1) / part1 + 1;
        auto num_part2 = (ap->dim4._2 - 1) / part2 + 1;

        LogAll(log_dbg, "p0:", ap->p0, " p1:", ap->p1, " p2:", ap->p2);
        LogAll(log_dbg, "num_part0:", num_part0, " num_part1:", num_part1, " num_part2:", num_part2);

        size_t block_stride1 = ap->p0, block_stride2 = block_stride1 * ap->p1;

        LogAll(log_dbg, "stride1:", ap->stride4._1, " stride2:", ap->stride4._2);
        LogAll(log_dbg, "blockstride1:", block_stride1, " blockstride2:", block_stride2);

        auto buffer_size = part0 * part1 * part2;
        LogAll(log_dbg, "buffer size:", buffer_size);
        auto quant_buffer = new Quant[buffer_size]();

        q = mem::CreateHostSpaceAndMemcpyFromDevice(d_q, len);

        Index<3>::idx_t part_dims{part0, part1, part2};
        Index<3>::idx_t block_strides{1, (int)block_stride1, (int)block_stride2};
        Index<3>::idx_t global_strides{1, (int)ap->stride4._1, (int)ap->stride4._2};

        for (auto pk = 0; pk < num_part2; pk++) {
            for (auto pj = 0; pj < num_part1; pj++) {
                for (auto pi = 0; pi < num_part0; pi++) {
                    auto start = pk * part2 * ap->stride4._2 + pj * part1 * ap->stride4._1 + pi * part0;
                    CopyToBuffer_3D(quant_buffer, q, start, part_dims, block_strides, global_strides);
                    lossless::interface::HuffmanEncodeWithTree_3D<Quant, Huff>(
                        Index<3>::idx_t{pi, pj, pk}, ap->c_huff_base, quant_buffer, buffer_size, ap->dict_size);
                }
            }
        }

        delete[] quant_buffer;
        delete[] q;

        hipFree(d_q);
        exit(0);
    }

    std::tie(n_bits, n_uInt, huffman_metadata_size, nvcomp_in_use) = lossless::interface::HuffmanEncode<Quant, Huff>(
        ap->c_huff_base, d_q, len, ap->huffman_chunk, ap->to_nvcomp, ap->dict_size, ap->export_codebook);

    LogAll(log_dbg, "to store Huffman encoded quant.code (default)");

    hipFree(d_q);
}

template <bool If_FP, int DataByte, int QuantByte, int HuffByte>
void cusz::interface::Decompress(
    argpack* ap,
    int&     nnz_outlier,
    size_t&  total_bits,
    size_t&  total_uInt,
    size_t&  huffman_metadata_size,
    bool     nvcomp_in_use)
{
    using Data  = typename DataTrait<If_FP, DataByte>::Data;
    using Quant = typename QuantTrait<QuantByte>::Quant;
    using Huff  = typename HuffTrait<HuffByte>::Huff;

    cout << "test extract" << endl;

    auto m   = ::cusz::impl::GetEdgeOfReinterpretedSquare(ap->len);
    auto mxm = m * m;

    LogAll(log_info, "invoke unzip");

    Quant* xq;
    // step 1: read from filesystem or do Huffman decoding to get quant code
    if (ap->skip_huffman) {
        LogAll(log_info, "load quant.code from filesystem");
        xq = io::ReadBinaryToNewArray<Quant>(ap->x_fi_q, ap->len);
    }
    else {
        LogAll(log_info, "Huffman decode -> quant.code");
        xq = lossless::interface::HuffmanDecode<Quant, Huff>(
            ap->cx_path2file, ap->len, ap->huffman_chunk, total_uInt, nvcomp_in_use, ap->dict_size);
        if (ap->verify_huffman) {
            LogAll(log_warn, "Verifying Huffman is temporarily disabled in this version (2021 Week 3");
            /*
            // TODO check in argpack
            if (ap->x_fi_origin == "") {
                cerr << log_err << "use \"--origin /path/to/origin_data\" to specify the original datum." << endl;
                exit(-1);
            }
            cout << log_info << "Verifying Huffman codec..." << endl;
            ::cusz::impl::VerifyHuffman<Data, Quant>(ap->x_fi_origin, len, xq, ap->huffman_chunk, dims, eb_variants);
             */
        }
    }
    auto d_xq = mem::CreateDeviceSpaceAndMemcpyFromHost(xq, ap->len);

    auto d_outlier = mem::CreateCUDASpace<Data>(mxm);
    ::cusz::impl::ScatterFromCSR<Data>(d_outlier, mxm, m /*lda*/, m /*m*/, m /*n*/, &nnz_outlier, &ap->x_fi_outlier);

    // TODO merge d_outlier and d_data
    auto d_xdata = mem::CreateCUDASpace<Data>(ap->len);

    {
        // temporary
        if (ap->ndim == 1) {
            LorenzoNdConfig<1, Data, workflow::unzip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            x_lorenzo_1d1l<Data, Quant>
                <<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.x_ctx, d_xdata, d_outlier, d_xq);
        }
        if (ap->ndim == 2) {
            LorenzoNdConfig<2, Data, workflow::unzip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            x_lorenzo_2d1l<Data, Quant>
                <<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.x_ctx, d_xdata, d_outlier, d_xq);
        }
        else if (ap->ndim == 3) {
            LorenzoNdConfig<3, Data, workflow::unzip> lc(ap->dim4, ap->stride4, ap->nblk4, ap->radius, ap->eb);
            x_lorenzo_3d1l<Data, Quant>
                <<<lc.cfg.Dg, lc.cfg.Db, lc.cfg.Ns, lc.cfg.S>>>(lc.x_ctx, d_xdata, d_outlier, d_xq);
        }
        HANDLE_ERROR(hipDeviceSynchronize());
    }

    auto xdata = mem::CreateHostSpaceAndMemcpyFromDevice(d_xdata, ap->len);

    LogAll(log_info, "reconstruct error-bounded datum");

    size_t archive_bytes = 0;
    // TODO huffman chunking metadata
    if (not ap->skip_huffman)
        archive_bytes += total_uInt * sizeof(Huff)  // Huffman coded
                         + huffman_metadata_size;   // chunking metadata and reverse codebook
    else
        archive_bytes += ap->len * sizeof(Quant);
    archive_bytes += nnz_outlier * (sizeof(Data) + sizeof(int)) + (m + 1) * sizeof(int);

    // TODO g++ and clang++ use mangled type_id name, add macro
    // https://stackoverflow.com/a/4541470/8740097
    auto demangle = [](const char* name) {
        int   status = -4;
        char* res    = abi::__cxa_demangle(name, nullptr, nullptr, &status);

        const char* const demangled_name = (status == 0) ? res : name;
        string            ret_val(demangled_name);
        free(res);
        return ret_val;
    };

    if (ap->skip_huffman) {
        cout << log_info << "dtype is \""         //
             << demangle(typeid(Data).name())     // demangle
             << "\", and quant. code type is \""  //
             << demangle(typeid(Quant).name())    // demangle
             << "\"; a CR of no greater than "    //
             << (sizeof(Data) / sizeof(Quant)) << " is expected when Huffman codec is skipped." << endl;
    }

    if (ap->pre_binning) cout << log_info << "Because of 2x2->1 binning, extra 4x CR is added." << endl;

    // TODO move CR out of VerifyData
    if (ap->x_fi_origin != "") {
        LogAll(log_info, "load the original datum for comparison");

        auto odata = io::ReadBinaryToNewArray<Data>(ap->x_fi_origin, ap->len);
        analysis::VerifyData(&ap->stat, xdata, odata, ap->len);
        analysis::PrintMetrics<Data>(&ap->stat, false, ap->eb, archive_bytes, ap->pre_binning ? 4 : 1);

        delete[] odata;
    }
    LogAll(log_info, "output:", ap->cx_path2file + ".szx");

    if (ap->skip_writex)
        io::WriteArrayToBinary(ap->x_fo_xd, xdata, ap->len);
    else {
        LogAll(log_dbg, "skipped writing unzipped to filesystem");
    }

    // clean up
    delete[] xdata;
    delete[] xq;
    hipFree(d_xdata);
    hipFree(d_outlier);
    hipFree(d_xq);
}

typedef struct DataPack<float> adp_f32_t;
namespace szin = cusz::interface;

// TODO top-level instantiation really reduce compilation time?
// clang-format off
template void szin::Compress<true, 4, 1, 4>(argpack*, adp_f32_t*, int&, size_t&, size_t&, size_t&, bool&);
template void szin::Compress<true, 4, 1, 8>(argpack*, adp_f32_t*, int&, size_t&, size_t&, size_t&, bool&);
template void szin::Compress<true, 4, 2, 4>(argpack*, adp_f32_t*, int&, size_t&, size_t&, size_t&, bool&);
template void szin::Compress<true, 4, 2, 8>(argpack*, adp_f32_t*, int&, size_t&, size_t&, size_t&, bool&);

template void szin::Decompress<true, 4, 1, 4>(argpack*, int&, size_t&, size_t&, size_t&, bool);
template void szin::Decompress<true, 4, 1, 8>(argpack*, int&, size_t&, size_t&, size_t&, bool);
template void szin::Decompress<true, 4, 2, 4>(argpack*, int&, size_t&, size_t&, size_t&, bool);
template void szin::Decompress<true, 4, 2, 8>(argpack*, int&, size_t&, size_t&, size_t&, bool);
