/**
 * @file cusz_workflow.cu
 * @author Jiannan Tian
 * @brief Workflow of cuSZ.
 * @version 0.1
 * @date 2020-09-20
 * Created on: 2020-02-12
 *
 * @copyright (C) 2020 by Washington State University, The University of Alabama, Argonne National Laboratory
 * See LICENSE in top-level directory
 *
 */

#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <cxxabi.h>
#include <bitset>
#include <cstdlib>
#include <exception>
#include <iostream>
#include <typeinfo>

// #if __cplusplus >= 201103L

#include <type_traits>

#include "analysis_utils.hh"
#include "argparse.hh"
#include "autotune.hh"
#include "constants.hh"
#include "cusz_interface.cuh"
#include "dryrun.cuh"
#include "dualquant.cuh"
#include "gather_scatter.cuh"
#include "huff_interface.cuh"
#include "metadata.hh"
#include "type_trait.hh"
#include "utils/cuda_err.cuh"
#include "utils/cuda_mem.cuh"
#include "utils/format.hh"
#include "utils/io.hh"
#include "utils/verify.hh"

using std::cerr;
using std::cout;
using std::endl;
using std::string;

typedef std::tuple<size_t, size_t, size_t, bool> tuple3ul;

template <typename Data, typename Quant>
void cusz::impl::PdQ(Data* d_d, Quant* d_q, size_t* dims, double* eb_variants)
{
    auto  d_dims        = mem::CreateDeviceSpaceAndMemcpyFromHost(dims, 16);
    auto  d_eb_variants = mem::CreateDeviceSpaceAndMemcpyFromHost(eb_variants, 4);
    void* args[]        = {&d_d, &d_q, &d_dims, &d_eb_variants};

    if (dims[nDIM] == 1) {
        static const int B = MetadataTrait<1>::Block;

        dim3 block_num(dims[nBLK0]);
        dim3 thread_num(B);
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::c_lorenzo_1d1l<Data, Quant>,  //
            block_num, thread_num, args, 0, nullptr);
    }
    else if (dims[nDIM] == 2) {
        static const int B = MetadataTrait<2>::Block;

        dim3 block_num(dims[nBLK0], dims[nBLK1]);
        dim3 thread_num(B, B);
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::c_lorenzo_2d1l<Data, Quant>,  //
            block_num, thread_num, args, (B + 1) * (B + 1) * sizeof(Data), nullptr);
    }
    else if (dims[nDIM] == 3) {
        static const int B = MetadataTrait<3>::Block;

        dim3 block_num(dims[nBLK0], dims[nBLK1], dims[nBLK2]);
        dim3 thread_num(B, B, B);
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::c_lorenzo_3d1l<Data, Quant>,  //
            block_num, thread_num, args, (B + 1) * (B + 1) * (B + 1) * sizeof(Data), nullptr);
    }
    HANDLE_ERROR(hipDeviceSynchronize());
}

template <typename Data, typename Quant>
void cusz::impl::ReversedPdQ(Data* d_xd, Quant* d_q, Data* d_outlier, size_t* dims, double _2eb)
{
    auto  d_dims = mem::CreateDeviceSpaceAndMemcpyFromHost(dims, 16);
    void* args[] = {&d_xd, &d_outlier, &d_q, &d_dims, &_2eb};

    if (dims[nDIM] == 1) {
        static const int p = MetadataTrait<1>::Block;

        dim3 thread_num(p);
        dim3 block_num((dims[nBLK0] - 1) / p + 1);
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::x_lorenzo_1d1l<Data, Quant>, block_num, thread_num, args, 0, nullptr);
    }
    else if (dims[nDIM] == 2) {
        const static size_t p = MetadataTrait<2>::Block;

        dim3 thread_num(p, p);
        dim3 block_num(
            (dims[nBLK0] - 1) / p + 1,   //
            (dims[nBLK1] - 1) / p + 1);  //
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::x_lorenzo_2d1l<Data, Quant>, block_num, thread_num, args, 0, nullptr);
    }
    else if (dims[nDIM] == 3) {
        const static size_t p = MetadataTrait<3>::Block;

        dim3 thread_num(p, p, p);
        dim3 block_num(
            (dims[nBLK0] - 1) / p + 1,   //
            (dims[nBLK1] - 1) / p + 1,   //
            (dims[nBLK2] - 1) / p + 1);  //
        hipLaunchKernel(
            (void*)cusz::predictor_quantizer::x_lorenzo_3d1l<Data, Quant>, block_num, thread_num, args, 0, nullptr);
    }
    else {
        cerr << log_err << "no 4D" << endl;
    }
    hipDeviceSynchronize();

    hipFree(d_dims);
}

template <typename Data, typename Quant>
void cusz::impl::VerifyHuffman(
    string const& fi,
    size_t        len,
    Quant*        xq,
    int           chunk_size,
    size_t*       dims,
    double*       eb_variants)
{
    // TODO error handling from invalid read
    cout << log_info << "Redo PdQ just to get quantization dump." << endl;

    auto  veri_data   = io::ReadBinaryToNewArray<Data>(fi, len);
    Data* veri_d_data = mem::CreateDeviceSpaceAndMemcpyFromHost(veri_data, len);
    auto  veri_d_q    = mem::CreateCUDASpace<Quant>(len);
    PdQ(veri_d_data, veri_d_q, dims, eb_variants);

    auto veri_q = mem::CreateHostSpaceAndMemcpyFromDevice(veri_d_q, len);

    auto count = 0;
    for (auto i = 0; i < len; i++)
        if (xq[i] != veri_q[i]) count++;
    if (count != 0)
        cerr << log_err << "percentage of not being equal: " << count / (1.0 * len) << "\n";
    else
        cout << log_info << "Decoded correctly." << endl;

    if (count != 0) {
        // auto chunk_size = ap->huffman_chunk;
        auto n_chunk = (len - 1) / chunk_size + 1;
        for (auto c = 0; c < n_chunk; c++) {
            auto chunk_id_printed   = false;
            auto prev_point_printed = false;
            for (auto i = 0; i < chunk_size; i++) {
                auto idx = i + c * chunk_size;
                if (idx >= len) break;
                if (xq[idx] != xq[idx]) {
                    if (not chunk_id_printed) {
                        cerr << "chunk id: " << c << "\t";
                        cerr << "start@ " << c * chunk_size << "\tend@ " << (c + 1) * chunk_size - 1 << endl;
                        chunk_id_printed = true;
                    }
                    if (not prev_point_printed) {
                        if (idx != c * chunk_size) {  // not first point
                            cerr << "PREV-idx:" << idx - 1 << "\t" << xq[idx - 1] << "\t" << xq[idx - 1] << endl;
                        }
                        else {
                            cerr << "wrong at first point!" << endl;
                        }
                        prev_point_printed = true;
                    }
                    cerr << "idx:" << idx << "\tdecoded: " << xq[idx] << "\tori: " << xq[idx] << endl;
                }
            }
        }
    }

    hipFree(veri_d_q);
    hipFree(veri_d_data);
    delete[] veri_q;
    delete[] veri_data;
    // end of if count
}

// clang-format off
template <bool If_FP, int DataByte, int QuantByte, int HuffByte>
void cusz::interface::Compress(
    argpack* ap,
    struct DataPack<typename DataTrait<If_FP, DataByte>::Data>* adp,
    size_t*  dims,
    double*  eb_variants,
    int&     nnz_outlier,
    size_t&  n_bits,
    size_t&  n_uInt,
    size_t&  huffman_metadata_size,
    bool&    nvcomp_in_use)
{
    // clang-format on
    using Data  = typename DataTrait<If_FP, DataByte>::Data;
    using Quant = typename QuantTrait<QuantByte>::Quant;
    using Huff  = typename HuffTrait<HuffByte>::Huff;

    // TODO to use a struct
    // TODO already calculated outside in main()
    size_t len = dims[LEN];

    auto data   = adp->data;
    auto d_data = adp->d_data;
    auto m      = adp->m;
    auto mxm    = adp->mxm;

    if (ap->to_dryrun) {
        logall(log_info, "invoke dry-run");
        DryRun(ap, data, d_data, ap->cx_path2file, dims, eb_variants);
        hipHostFree(data);
        hipFree(d_data);
        exit(0);
    }
    logall(log_info, "invoke zipping");

    auto d_q = mem::CreateCUDASpace<Quant>(len);  // quant. code is not needed for dry-run

    // prediction-quantization
    ::cusz::impl::PdQ(d_data, d_q, dims, eb_variants);
    ::cusz::impl::PruneGatherAsCSR(d_data, mxm, m /*lda*/, m /*m*/, m /*n*/, nnz_outlier, &ap->c_fo_outlier);

    auto fmt_nnz = "(" + std::to_string(nnz_outlier / 1.0 / len * 100) + "%)";
    logall(log_info, "nnz/#outlier:", nnz_outlier, fmt_nnz, "saved");
    hipFree(d_data);  // ad-hoc, release memory for large dataset

    Quant* q;
    if (ap->skip_huffman) {
        q = mem::CreateHostSpaceAndMemcpyFromDevice(d_q, len);
        io::WriteArrayToBinary(ap->c_fo_q, q, len);

        logall(log_info, "to store quant.code directly (Huffman enc skipped)");

        return;
    }

    // autotuning Huffman chunksize
    // subject to change, current `8*` is close to but may note deterministically optimal
    if (ap->autotune_huffman_chunk) {  //
        auto optimal_chunksize = 1;
        auto cuda_core_num     = cusz::tune::GetCUDACoreNum();
        auto cuda_thread_num   = 8 * cuda_core_num;  // empirical value

        while (optimal_chunksize * cuda_thread_num < len) optimal_chunksize *= 2;
        ap->huffman_chunk = optimal_chunksize;
    }

    std::tie(n_bits, n_uInt, huffman_metadata_size, nvcomp_in_use) =
        lossless::interface::HuffmanEncode<Quant, Huff>(ap->c_huff_base, d_q, len, ap->huffman_chunk, ap->to_nvcomp, dims[CAP]);

    logall(log_dbg, "to store Huffman encoded quant.code (default)");

    hipFree(d_q);
}

template <bool If_FP, int DataByte, int QuantByte, int HuffByte>
void cusz::interface::Decompress(
    argpack* ap,
    size_t*  dims,
    double*  eb_variants,
    int&     nnz_outlier,
    size_t&  total_bits,
    size_t&  total_uInt,
    size_t&  huffman_metadata_size,
    bool     nvcomp_in_use)
{
    using Data  = typename DataTrait<If_FP, DataByte>::Data;
    using Quant = typename QuantTrait<QuantByte>::Quant;
    using Huff  = typename HuffTrait<HuffByte>::Huff;

    auto dict_size = dims[CAP];
    auto len       = dims[LEN];
    auto m         = ::cusz::impl::GetEdgeOfReinterpretedSquare(len);
    auto mxm       = m * m;

    logall(log_info, "invoke unzip");

    Quant* xq;
    // step 1: read from filesystem or do Huffman decoding to get quant code
    if (ap->skip_huffman) {
        logall(log_info, "load quant.code from filesystem");
        xq = io::ReadBinaryToNewArray<Quant>(ap->x_fi_q, len);
    }
    else {
        logall(log_info, "Huffman decode -> quant.code");
        xq = lossless::interface::HuffmanDecode<Quant, Huff>(
            ap->cx_path2file, len, ap->huffman_chunk, total_uInt, nvcomp_in_use, dict_size);
        if (ap->verify_huffman) {
            // TODO check in argpack
            if (ap->x_fi_origin == "") {
                cerr << log_err << "use \"--origin /path/to/origin_data\" to specify the original datum." << endl;
                exit(-1);
            }
            cout << log_info << "Verifying Huffman codec..." << endl;
            ::cusz::impl::VerifyHuffman<Data, Quant>(ap->x_fi_origin, len, xq, ap->huffman_chunk, dims, eb_variants);
        }
    }
    auto d_xq = mem::CreateDeviceSpaceAndMemcpyFromHost(xq, len);

    auto d_outlier = mem::CreateCUDASpace<Data>(mxm);
    ::cusz::impl::ScatterFromCSR<Data>(d_outlier, mxm, m /*lda*/, m /*m*/, m /*n*/, &nnz_outlier, &ap->x_fi_outlier);

    // TODO merge d_outlier and d_data
    auto d_xdata = mem::CreateCUDASpace<Data>(len);
    ::cusz::impl::ReversedPdQ(d_xdata, d_xq, d_outlier, dims, eb_variants[EBx2]);
    auto xdata = mem::CreateHostSpaceAndMemcpyFromDevice(d_xdata, len);

    logall(log_info, "reconstruct error-bounded datum");

    size_t archive_bytes = 0;
    // TODO huffman chunking metadata
    if (not ap->skip_huffman)
        archive_bytes += total_uInt * sizeof(Huff)  // Huffman coded
                         + huffman_metadata_size;   // chunking metadata and reverse codebook
    else
        archive_bytes += len * sizeof(Quant);
    archive_bytes += nnz_outlier * (sizeof(Data) + sizeof(int)) + (m + 1) * sizeof(int);

    // TODO g++ and clang++ use mangled type_id name, add macro
    // https://stackoverflow.com/a/4541470/8740097
    auto demangle = [](const char* name) {
        int   status = -4;
        char* res    = abi::__cxa_demangle(name, nullptr, nullptr, &status);

        const char* const demangled_name = (status == 0) ? res : name;
        string            ret_val(demangled_name);
        free(res);
        return ret_val;
    };

    if (ap->skip_huffman) {
        cout << log_info << "dtype is \""         //
             << demangle(typeid(Data).name())     // demangle
             << "\", and quant. code type is \""  //
             << demangle(typeid(Quant).name())    // demangle
             << "\"; a CR of no greater than "    //
             << (sizeof(Data) / sizeof(Quant)) << " is expected when Huffman codec is skipped." << endl;
    }

    if (ap->pre_binning) cout << log_info << "Because of 2x2->1 binning, extra 4x CR is added." << endl;

    // TODO move CR out of VerifyData
    if (ap->x_fi_origin != "") {
        logall(log_info, "load the original datum for comparison");

        auto odata = io::ReadBinaryToNewArray<Data>(ap->x_fi_origin, len);
        analysis::VerifyData(&ap->stat, xdata, odata, len);
        analysis::PrintMetrics<Data>(&ap->stat, false, eb_variants[EB], archive_bytes, ap->pre_binning ? 4 : 1);

        delete[] odata;
    }
    logall(log_info, "output:", ap->cx_path2file + ".szx");

    if (!ap->skip_writex)
        io::WriteArrayToBinary(ap->x_fo_xd, xdata, len);
    else {
        logall(log_dbg, "skipped writing unzipped to filesystem");
    }

    // clean up
    delete[] xdata;
    delete[] xq;
    hipFree(d_xdata);
    hipFree(d_outlier);
    hipFree(d_xq);
}

typedef struct DataPack<float> adp_f32_t;
namespace szin = cusz::interface;

template void szin::Compress<true, 4, 1, 4>(argpack*, adp_f32_t*, size_t*, FP8*, int&, size_t&, size_t&, size_t&, bool&);
template void szin::Compress<true, 4, 1, 8>(argpack*, adp_f32_t*, size_t*, FP8*, int&, size_t&, size_t&, size_t&, bool&);
template void szin::Compress<true, 4, 2, 4>(argpack*, adp_f32_t*, size_t*, FP8*, int&, size_t&, size_t&, size_t&, bool&);
template void szin::Compress<true, 4, 2, 8>(argpack*, adp_f32_t*, size_t*, FP8*, int&, size_t&, size_t&, size_t&, bool&);

template void szin::Decompress<true, 4, 1, 4>(argpack*, size_t*, FP8*, int&, size_t&, size_t&, size_t&, bool);
template void szin::Decompress<true, 4, 1, 8>(argpack*, size_t*, FP8*, int&, size_t&, size_t&, size_t&, bool);
template void szin::Decompress<true, 4, 2, 4>(argpack*, size_t*, FP8*, int&, size_t&, size_t&, size_t&, bool);
template void szin::Decompress<true, 4, 2, 8>(argpack*, size_t*, FP8*, int&, size_t&, size_t&, size_t&, bool);
